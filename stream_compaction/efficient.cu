#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"
#include ""
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "thrust.h"

#define NUM_BANKS 16
#define LOG_NUM_BANKS 4
#define CONFLICT_FREE_OFFSET(n) \
    ((n) >> NUM_BANKS + (n) >> (2 * LOG_NUM_BANKS))
#define BLOCKSIZE_SHARED 64

namespace StreamCompaction {
	namespace Efficient {
		using StreamCompaction::Common::PerformanceTimer;
		PerformanceTimer& timer()
		{
			static PerformanceTimer timer;
			return timer;
		}

		__global__ void kernUpSweep(int n, int d, int *odata) {
			int index = (blockDim.x * blockIdx.x) + threadIdx.x;
			if (index >= n) {
				return;
			}
			int step = 1 << (d + 1);
			if (index % step == 0) {
				odata[index + step - 1] += odata[index + (1 << d) - 1];
			}
		}

		__global__ void kernDownSweep(int n, int d, int *odata) {
			int index = (blockDim.x * blockIdx.x) + threadIdx.x;
			if (index >= n) {
				return;
			}
			int step = 1 << (d + 1);
			if (index % step == 0) {
				int t = odata[index + (1 << d) - 1];
				odata[index + (1 << d) - 1] = odata[index + step - 1];
				odata[index + step - 1] += t;
			}
		}



		/**
		* Performs prefix-sum (aka scan) on idata, storing the result into odata.
		*/
		void nonoptscan(int n, int *odata, const int *idata) {
			// TODO
			int upLimit = ilog2ceil(n);
			int len = 1 << upLimit;

			int blockSize = 1024;
			dim3 threadsPerBlock(blockSize);
			dim3 blocksPerGrid((len + blockSize - 1) / blockSize);

			int *dev_data;
			hipMalloc((void**)&dev_data, len * sizeof(int));
			checkCUDAError("hipMalloc dev_data failed!");
			hipMemcpy(dev_data, idata, len * sizeof(int), hipMemcpyHostToDevice);
			checkCUDAError("hipMemcpy dev_data failed!");



			timer().startGpuTimer();
			for (int d = 0; d <= upLimit - 1; d++) {
				kernUpSweep << <blocksPerGrid, threadsPerBlock >> > (len, d, dev_data);
				checkCUDAError("kernUpSweep failed!");
			}

			hipMemset(&dev_data[len - 1], 0, sizeof(int));
			checkCUDAError("hipMemcpy set last one to be zero failed!");

			for (int d = upLimit - 1; d >= 0; d--) {
				kernDownSweep << <blocksPerGrid, threadsPerBlock >> > (len, d, dev_data);
				checkCUDAError("kernDownSweep failed!");
			}
			timer().endGpuTimer();
			hipMemcpy(odata, dev_data, len * sizeof(int), hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpy dev_data failed!");
			hipFree(dev_data);
			hipDeviceSynchronize();
		}

		__global__ void kernOptUpSweep(int n, int d, int *odata) {
			int index = (blockDim.x * blockIdx.x) + threadIdx.x;
			if (index >= n) {
				return;
			}
			int step = 1 << (d + 1);
			odata[index * step + step - 1] += odata[index * step + (1 << d) - 1];
		}

		__global__ void kernOptDownSweep(int n, int d, int *odata) {
			int index = (blockDim.x * blockIdx.x) + threadIdx.x;
			if (index >= n) {
				return;
			}
			int step = 1 << (d + 1);
			int t = odata[index*step + (1 << d) - 1];
			odata[index * step + (1 << d) - 1] = odata[index * step + step - 1];
			odata[index * step + step - 1] += t;
		}

		void scan(int n, int *odata, const int *idata) {
			int upLimit = ilog2ceil(n);
			int len = 1 << upLimit;

			int blockSize = 1024;
			dim3 threadsPerBlock(blockSize);
			dim3 blocksPerGrid((len + blockSize - 1) / blockSize);

			int *dev_data;
			hipMalloc((void**)&dev_data, len * sizeof(int));
			checkCUDAError("hipMalloc dev_data failed!");
			hipMemcpy(dev_data, idata, len * sizeof(int), hipMemcpyHostToDevice);
			checkCUDAError("hipMemcpy dev_data failed!");



			timer().startGpuTimer();
			for (int d = 0; d <= upLimit - 1; d++) {
				int step = 1 << (d + 1);
				int tempLen = len / step;
				blocksPerGrid = dim3((tempLen + blockSize) / blockSize);
				kernOptUpSweep << <blocksPerGrid, threadsPerBlock >> > (tempLen, d, dev_data);
				checkCUDAError("kernUpSweep failed!");
			}

			hipMemset(&dev_data[len - 1], 0, sizeof(int));
			checkCUDAError("hipMemcpy set last one to be zero failed!");

			for (int d = upLimit - 1; d >= 0; d--) {
				int step = 1 << (d + 1);
				int tempLen = len / step;
				blocksPerGrid = dim3((tempLen + blockSize) / blockSize);
				kernOptDownSweep << <blocksPerGrid, threadsPerBlock >> > (tempLen, d, dev_data);
				checkCUDAError("kernDownSweep failed!");
			}
			timer().endGpuTimer();
			hipMemcpy(odata, dev_data, len * sizeof(int), hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpy dev_data failed!");
			hipFree(dev_data);
			hipDeviceSynchronize();
		}

		void gpuScan(int n, int *data) {
			int upLimit = ilog2ceil(n);
			int blockSize = 1024;
			dim3 threadsPerBlock(blockSize);
			dim3 blocksPerGrid((n + blockSize - 1) / blockSize);

			for (int d = 0; d <= upLimit - 1; d++) {
				kernUpSweep << <blocksPerGrid, threadsPerBlock >> > (n, d, data);
				checkCUDAError("kernUpSweep failed!");
			}

			hipMemset(&data[n - 1], 0, sizeof(int));
			checkCUDAError("hipMemcpy set last one to be zero failed!");

			for (int d = upLimit - 1; d >= 0; d--) {
				kernDownSweep << <blocksPerGrid, threadsPerBlock >> > (n, d, data);
				checkCUDAError("kernDownSweep failed!");
			}
		}

		__global__ void kernSharedMemoryScan(int n, int *odata, const int *idata, int *blockData) {
			int index = (blockDim.x * blockIdx.x) + threadIdx.x;
			if (index >= n) {
				return;
			}
			extern __shared__ int temp[BLOCKSIZE_SHARED + NUM_BANKS];
			int thid = threadIdx.x;
			int offset = 1;
			int stride = blockIdx.x * blockDim.x;
			int ai = thid << 1;
			int bi = ai + 1;
			int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
			int bankOffsetB = CONFLICT_FREE_OFFSET(bi);
			temp[ai + bankOffsetA] = idata[ai + stride];
			temp[bi + bankOffsetB] = idata[bi + stride];
			for (int d = BLOCKSIZE_SHARED >> 1; d > 0; d >>= 1)
			{
				__syncthreads();
				if (thid < d)
				{
					int ai = offset*(2 * thid + 1) - 1;
					int bi = offset*(2 * thid + 2) - 1;
					ai += CONFLICT_FREE_OFFSET(ai);
					bi += CONFLICT_FREE_OFFSET(bi);
					temp[bi] += temp[ai];
				}
				offset *= 2;
			}
			__syncthreads();
			if (thid != 0) {
				blockData[blockIdx.x] = temp[BLOCKSIZE_SHARED - 1 + CONFLICT_FREE_OFFSET(BLOCKSIZE_SHARED - 1)];
				temp[BLOCKSIZE_SHARED - 1 + CONFLICT_FREE_OFFSET(BLOCKSIZE_SHARED - 1)] = 0;
			}
			for (int d = 1; d < BLOCKSIZE_SHARED; d *= 2) // traverse down tree & build scan
			{
				offset >>= 1;
				__syncthreads();
				if (thid < d)
				{
					int ai = offset*(2 * thid + 1) - 1;
					int bi = offset*(2 * thid + 2) - 1;
					ai += CONFLICT_FREE_OFFSET(ai);
					bi += CONFLICT_FREE_OFFSET(bi);
					float t = temp[ai];
					temp[ai] = temp[bi];
					temp[bi] += t;
				}
			}
			ai = thid << 1;
			bi = ai + 1;
			odata[ai + stride] = temp[ai + bankOffsetA];
			odata[bi + stride] = temp[bi + bankOffsetB];
		}

		__global__ void kernAddBlockData(int n, const int *idata, int *odata, const int *blockData) {
			int index = (blockDim.x * blockIdx.x) + threadIdx.x;
			if (index >= n) {
				return;
			}
			int bid = blockIdx.x;
			if (bid > 0) {
				odata[index] = idata[index] + blockData[index];
			}
			else {
				odata[index] = idata[index];
			}
		}

		void sharedMemoryScan(int n, int *odata, const int *idata) {
			int upLimit = ilog2ceil(n);
			int len = 1 << upLimit;

			int blockSize = 64;
			dim3 threadsPerBlock(BLOCKSIZE_SHARED / 2);
			dim3 blocksPerGrid((n + BLOCKSIZE_SHARED - 1) / blockSize);


			int alen = ((n + BLOCKSIZE_SHARED - 1) / BLOCKSIZE_SHARED);
			dim3 threadsPerBlock_Block(BLOCKSIZE_SHARED / 2);
			dim3 blocksPerGrid_Block((alen + BLOCKSIZE_SHARED - 1) / BLOCKSIZE_SHARED);
			int *dev_idata;
			int *dev_odata;
			int *dev_blockData;
			int *dev_blockSum;
			hipMalloc((void**)&dev_idata, len * sizeof(int));
			checkCUDAError("hipMalloc dev_idata failed!");
			hipMalloc((void**)&dev_odata, len * sizeof(int));
			checkCUDAError("hipMalloc dev_odata failed!");
			hipMalloc((void**)&dev_blockData, alen * sizeof(int));
			checkCUDAError("hipMalloc dev_blockData failed!");
			hipMalloc((void**)&dev_blockSum, alen * sizeof(int));
			checkCUDAError("hipMalloc dev_blockSum failed!");
			hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
			checkCUDAError("hipMemcpy dev_idata failed!");

			int *host_aux, *host_aux_sum;
			hipHostMalloc((void**)&host_aux, alen * sizeof(int)); // Use Pin-Memory to ensure the maximum memory speed
			hipHostMalloc((void**)&host_aux_sum, alen * sizeof(int));

			timer().startGpuTimer();
			kernSharedMemoryScan << <blocksPerGrid, threadsPerBlock >> > (len, dev_odata, dev_idata, dev_blockData);
			checkCUDAError("kernSharedMemoryScan failed!");
			hipMemcpy(host_aux, dev_blockData, alen * sizeof(int), hipMemcpyDeviceToHost);
			thrust::inclusive_scan(host_aux, host_aux + alen, host_aux_sum);
			hipMemcpy(dev_blockSum, host_aux_sum, alen * sizeof(int), hipMemcpyHostToDevice);
			kernAddBlockData << <blocksPerGrid, BLOCKSIZE_SHARED >> > (n, dev_odata, dev_idata, dev_blockSum);
			checkCUDAError("kernSharedMemoryScan failed!");
			timer().endGpuTimer();
			hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpy dev_odata failed!");

			hipFree(dev_idata);
			hipFree(dev_odata);
			hipFree(dev_blockData);
			hipFree(dev_blockSum);
			hipDeviceSynchronize();
			hipHostFree(host_aux);
			hipHostFree(host_aux_sum);
		}



		/**
		* Performs stream compaction on idata, storing the result into odata.
		* All zeroes are discarded.
		*
		* @param n      The number of elements in idata.
		* @param odata  The array into which to store elements.
		* @param idata  The array of elements to compact.
		* @returns      The number of elements remaining after compaction.
		*/
		int compact(int n, int *odata, const int *idata) {
			int upLimit = ilog2ceil(n);
			int len = 1 << upLimit;
			int blockSize = 1024;
			dim3 threadsPerBlock(blockSize);
			dim3 blocksPerGrid((len + blockSize - 1) / blockSize);
			int *dev_odata;
			int *dev_idata;
			int *dev_bools;
			int *dev_indices;

			hipMalloc((void**)&dev_odata, sizeof(int) * len);
			checkCUDAError("hipMalloc dev_odata failed!");
			hipMalloc((void**)&dev_idata, sizeof(int) * len);
			checkCUDAError("hipMalloc dev_idata failed!");
			hipMalloc((void**)&dev_bools, sizeof(int) * len);
			checkCUDAError("hipMalloc dev_bools failed!");
			hipMalloc((void**)&dev_indices, sizeof(int) * len);
			checkCUDAError("hipMalloc dev_indices failed!");
			hipMemcpy(dev_idata, idata, sizeof(int) * n, hipMemcpyHostToDevice);
			checkCUDAError("hipMemcpy dev_idata failed!");

			timer().startGpuTimer();

			StreamCompaction::Common::kernMapToBoolean << <blocksPerGrid, threadsPerBlock >> > (len, dev_bools, dev_idata);
			checkCUDAError("kernMaptoBoolean failed!");

			hipMemcpy(dev_indices, dev_bools, sizeof(int) * len, hipMemcpyDeviceToDevice);
			hipMemset(&dev_odata[n], 0, sizeof(int) * (len - n));
			gpuScan(len, dev_indices);

			StreamCompaction::Common::kernScatter << <blocksPerGrid, threadsPerBlock >> > (len, dev_odata, dev_idata, dev_bools, dev_indices);
			checkCUDAError("kernScatter failed!");

			timer().endGpuTimer();
			hipMemcpy(odata, dev_odata, sizeof(int) * len, hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpy odata failed!");

			hipFree(dev_odata);
			hipFree(dev_idata);
			hipFree(dev_indices);
			hipFree(dev_bools);
			int count = 0;
			while (odata[count] != 0) {
				count++;
			}
			return count;
		}
	}
}
