#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"
#include ""
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "thrust.h"

#define NUM_BANKS 16
#define LOG_NUM_BANKS 4
#define CONFLICT_FREE_OFFSET(n) \
    ((n) >> NUM_BANKS + (n) >> (2 * LOG_NUM_BANKS))

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

		__global__ void kernUpSweep(int n, int d, int *odata) {
			int index = (blockDim.x * blockIdx.x) + threadIdx.x;
			if (index >= n) {
				return;
			}
			int step = 1 << (d + 1);
			if (index % step == 0) {
				odata[index + step - 1] += odata[index + (1 << d) - 1];
			}
		}

		__global__ void kernDownSweep(int n, int d, int *odata) {
			int index = (blockDim.x * blockIdx.x) + threadIdx.x;
			if (index >= n) {
				return;
			}
			int step = 1 << (d + 1);
			if (index % step == 0) {
				int t = odata[index + (1 << d) - 1];
				odata[index + (1 << d) - 1] = odata[index + step - 1];
				odata[index + step - 1] += t;
			}
		}



        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void nonoptscan(int n, int *odata, const int *idata) {         
            // TODO
			int upLimit = ilog2ceil(n);
			int len = 1 << upLimit;

			int blockSize = 1024;
			dim3 threadsPerBlock(blockSize);
			dim3 blocksPerGrid((len + blockSize - 1) / blockSize);

			int *dev_data;
			hipMalloc((void**)&dev_data, len * sizeof(int));
			checkCUDAError("hipMalloc dev_data failed!");
			hipMemcpy(dev_data, idata, len * sizeof(int), hipMemcpyHostToDevice);
			checkCUDAError("hipMemcpy dev_data failed!");
			
			

			timer().startGpuTimer();
			for (int d = 0; d <= upLimit - 1; d++) {
				kernUpSweep << <blocksPerGrid, threadsPerBlock >> > (len, d, dev_data);
				checkCUDAError("kernUpSweep failed!");
			}

			hipMemset(&dev_data[len - 1], 0, sizeof(int));
			checkCUDAError("hipMemcpy set last one to be zero failed!");
			
			for (int d = upLimit - 1; d >= 0; d--) {
				kernDownSweep << <blocksPerGrid, threadsPerBlock >> > (len, d, dev_data);
				checkCUDAError("kernDownSweep failed!");
			}
			timer().endGpuTimer();
			hipMemcpy(odata, dev_data, len * sizeof(int), hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpy dev_data failed!");
			hipFree(dev_data);
			hipDeviceSynchronize();            
        }

		__global__ void kernOptUpSweep(int n, int d, int *odata) {
			int index = (blockDim.x * blockIdx.x) + threadIdx.x;
			if (index >= n) {
				return;
			}
			int step = 1 << (d + 1);
			odata[index * step + step - 1] += odata[index * step + (1 << d) - 1];
		}

		__global__ void kernOptDownSweep(int n, int d, int *odata) {
			int index = (blockDim.x * blockIdx.x) + threadIdx.x;
			if (index >= n) {
				return;
			}
			int step = 1 << (d + 1);
			int t = odata[index*step + (1 << d) - 1];
			odata[index * step + (1 << d) - 1] = odata[index * step + step - 1];
			odata[index * step + step - 1] += t;
		}

		void scan(int n, int *odata, const int *idata) {
			int upLimit = ilog2ceil(n);
			int len = 1 << upLimit;

			int blockSize = 1024;
			dim3 threadsPerBlock(blockSize);
			dim3 blocksPerGrid((len + blockSize - 1) / blockSize);

			int *dev_data;
			hipMalloc((void**)&dev_data, len * sizeof(int));
			checkCUDAError("hipMalloc dev_data failed!");
			hipMemcpy(dev_data, idata, len * sizeof(int), hipMemcpyHostToDevice);
			checkCUDAError("hipMemcpy dev_data failed!");



			timer().startGpuTimer();
			for (int d = 0; d <= upLimit - 1; d++) {
				int step = 1 << (d + 1);
				int tempLen = len / step;
				blocksPerGrid = dim3((tempLen + blockSize) / blockSize);
				kernOptUpSweep << <blocksPerGrid, threadsPerBlock >> > (tempLen, d, dev_data);
				checkCUDAError("kernUpSweep failed!");
			}

			hipMemset(&dev_data[len - 1], 0, sizeof(int));
			checkCUDAError("hipMemcpy set last one to be zero failed!");

			for (int d = upLimit - 1; d >= 0; d--) {
				int step = 1 << (d + 1);
				int tempLen = len / step;
				blocksPerGrid = dim3((tempLen + blockSize) / blockSize);
				kernOptDownSweep << <blocksPerGrid, threadsPerBlock >> > (tempLen, d, dev_data);
				checkCUDAError("kernDownSweep failed!");
			}
			timer().endGpuTimer();
			hipMemcpy(odata, dev_data, len * sizeof(int), hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpy dev_data failed!");
			hipFree(dev_data);
			hipDeviceSynchronize();
		}

		void gpuScan(int n, int *data) {
			int upLimit = ilog2ceil(n);
			int blockSize = 1024;
			dim3 threadsPerBlock(blockSize);
			dim3 blocksPerGrid((n + blockSize - 1) / blockSize);

			for (int d = 0; d <= upLimit - 1; d++) {
				kernUpSweep << <blocksPerGrid, threadsPerBlock >> > (n, d, data);
				checkCUDAError("kernUpSweep failed!");
			}

			hipMemset(&data[n - 1], 0, sizeof(int));
			checkCUDAError("hipMemcpy set last one to be zero failed!");

			for (int d = upLimit - 1; d >= 0; d--) {
				kernDownSweep << <blocksPerGrid, threadsPerBlock >> > (n, d, data);
				checkCUDAError("kernDownSweep failed!");
			}
		}

		__global__ void kernSharedMemoryScan(int n, int *odata, const int *idata, int *blockData) {
			int index = (blockDim.x * blockIdx.x) + threadIdx.x;
			if (index >= n) {
				return;
			}
			extern __shared__ int temp[];
			int thid = threadIdx.x;
			int offset = 1;
			int stride = blockIdx.x * blockDim.x;
			int ai = thid;
			int bi = thid + (n / 2);
			int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
			int bankOffsetB = CONFLICT_FREE_OFFSET(bi);
			temp[ai + bankOffsetA] = idata[ai + stride];
			temp[bi + bankOffsetB] = idata[bi + stride];
			for (int d = n >> 1; d > 0; d >>= 1)
			{
				__syncthreads();
				if (thid < d)
				{
					int ai = offset*(2 * thid + 1) - 1;
					int bi = offset*(2 * thid + 2) - 1;
					ai += CONFLICT_FREE_OFFSET(ai);
					bi += CONFLICT_FREE_OFFSET(bi);
					temp[bi] += temp[ai];
				}
				offset *= 2;
			}
			if (thid == 0) { 
				blockData[blockIdx.x] = temp[n - 1 + CONFLICT_FREE_OFFSET(n - 1)];
				temp[n - 1 + CONFLICT_FREE_OFFSET(n - 1)] = 0; 
			}
			for (int d = 1; d < n; d *= 2) // traverse down tree & build scan
			{
				offset >>= 1;
				__syncthreads();
				if (thid < d)
				{
					int ai = offset*(2 * thid + 1) - 1;
					int bi = offset*(2 * thid + 2) - 1;
					ai += CONFLICT_FREE_OFFSET(ai);
					bi += CONFLICT_FREE_OFFSET(bi);
					float t = temp[ai];
					temp[ai] = temp[bi];
					temp[bi] += t;
				}
			}
			odata[ai + stride] = temp[ai + bankOffsetA];
			odata[bi + stride] = temp[bi + bankOffsetB];
		}

		__global__ void kernAddBlockData(int n, int *odata, const int *idata, const int *blockData) {
			int index = (blockDim.x * blockIdx.x) + threadIdx.x;
			if (index >= n) {
				return;
			}
			odata[index] = idata[index] + blockData[blockIdx.x];
		}

		void sharedMemoryScan(int n, int *odata, const int *idata) {
			int upLimit = ilog2ceil(n);
			int len = 1 << upLimit;

			int blockSize = 64;
			dim3 threadsPerBlock(blockSize);
			dim3 blocksPerGrid((len + blockSize - 1) / blockSize);
			int partSize = (len + blockSize - 1) / blockSize;
			int *dev_idata;
			int *dev_odata;
			int *dev_blockData;
			int *dev_blockSum;
			hipMalloc((void**)&dev_idata, len * sizeof(int));
			checkCUDAError("hipMalloc dev_idata failed!");
			hipMalloc((void**)&dev_odata, len * sizeof(int));
			checkCUDAError("hipMalloc dev_odata failed!");
			hipMalloc((void**)&dev_blockData, partSize * sizeof(int));
			checkCUDAError("hipMalloc dev_blockData failed!");
			hipMalloc((void**)&dev_blockSum, partSize * sizeof(int));
			checkCUDAError("hipMalloc dev_blockSum failed!");
			hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
			checkCUDAError("hipMemcpy dev_idata failed!");



			timer().startGpuTimer();
			kernSharedMemoryScan << <blocksPerGrid, threadsPerBlock >> > (len, dev_odata, dev_idata, dev_blockData);
			checkCUDAError("kernSharedMemoryScan failed!");
			//thrust::inclusive_scan(dev_blockData, dev_blockData + n, dev_blockSum);
			//kernAddBlockData << <blocksPerGrid, threadsPerBlock >> > (len, dev_odata, dev_idata, dev_blockSum);
			//checkCUDAError("kernSharedMemoryScan failed!");
			timer().endGpuTimer();
			hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpy dev_odata failed!");

			hipFree(dev_idata);
			hipFree(dev_odata);
			hipFree(dev_blockData);
			hipFree(dev_blockSum);
			hipDeviceSynchronize();
		}



        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
			int upLimit = ilog2ceil(n);
			int len = 1 << upLimit;
			int blockSize = 1024;
			dim3 threadsPerBlock(blockSize);
			dim3 blocksPerGrid((len + blockSize - 1) / blockSize);
			int *dev_odata;
			int *dev_idata;
			int *dev_bools;
			int *dev_indices;

			hipMalloc((void**)&dev_odata, sizeof(int) * len);
			checkCUDAError("hipMalloc dev_odata failed!");
			hipMalloc((void**)&dev_idata, sizeof(int) * len);
			checkCUDAError("hipMalloc dev_idata failed!");
			hipMalloc((void**)&dev_bools, sizeof(int) * len);
			checkCUDAError("hipMalloc dev_bools failed!");
			hipMalloc((void**)&dev_indices, sizeof(int) * len);
			checkCUDAError("hipMalloc dev_indices failed!");
			hipMemcpy(dev_idata, idata, sizeof(int) * n, hipMemcpyHostToDevice);
			checkCUDAError("hipMemcpy dev_idata failed!");

            timer().startGpuTimer();

			StreamCompaction::Common::kernMapToBoolean << <blocksPerGrid, threadsPerBlock >> > (len, dev_bools, dev_idata);
			checkCUDAError("kernMaptoBoolean failed!");

			hipMemcpy(dev_indices, dev_bools, sizeof(int) * len, hipMemcpyDeviceToDevice);
			hipMemset(&dev_odata[n], 0, sizeof(int) * (len - n));
			gpuScan(len, dev_indices);
			
			StreamCompaction::Common::kernScatter << <blocksPerGrid, threadsPerBlock >> > (len, dev_odata, dev_idata, dev_bools, dev_indices);
			checkCUDAError("kernScatter failed!");

			timer().endGpuTimer();
			hipMemcpy(odata, dev_odata, sizeof(int) * len, hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpy odata failed!");

			hipFree(dev_odata);
			hipFree(dev_idata);
			hipFree(dev_indices);
			hipFree(dev_bools);
			int count = 0;
			while (odata[count] != 0) { 
				count++; 
			}
            return count;
        }
    }
}
