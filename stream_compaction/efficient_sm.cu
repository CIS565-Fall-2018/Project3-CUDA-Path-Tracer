#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient_sm.h"

namespace StreamCompaction {
    namespace EfficientSM {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }


        __global__ void kernEfficientScan(int N, int *odata, int *idata){
            extern __shared__ int tmp[];
            int index = threadIdx.x;
            if (index >= N) return;

            int offset = 1;
            tmp[2 * index] = idata[2 * index];
            tmp[2 * index + 1] = idata[2 * index + 1];
            // up sweep
            for (int d = (N >> 1); d > 0; d >>= 1){
                __syncthreads();
                if (index < d) tmp[offset * (2 * index + 2) - 1] += tmp[offset * (2 * index + 1) - 1];
                offset <<= 1;
            }
            // clear last digit
            if (index == 0) tmp[N - 1] = 0;
            // down sweep
            for (int d = 1; d < N; d <<= 1){
                offset >>= 1;
                __syncthreads();
                if (index < d){
                    int t = tmp[offset * (2 * index + 1) - 1];
                    tmp[offset * (2 * index + 1) - 1] = tmp[offset * (2 * index + 2) - 1];
                    tmp[offset * (2 * index + 2) - 1] += t;
                }
            }
            __syncthreads();

            odata[2 * index] = tmp[2 * index];
            odata[2 * index + 1] = tmp[2 * index + 1];
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            int N = 1 << ilog2ceil(n);
            dim3 fullBlockPerGrid((N + blockSize - 1) / blockSize);
            int* dev_in, *dev_out;

            hipMalloc((void**) &dev_in, N * sizeof(int));
            checkCUDAError("hipMalloc dev_in failed");

            hipMalloc((void**) &dev_out, N * sizeof(int));
            checkCUDAError("hipMalloc dev_out failed");

            hipMemset(dev_out, 0, sizeof(int) * N);
            checkCUDAError("cuda Memset failed");

            hipMemcpy(dev_in, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy HostToDevice failed");

            timer().startGpuTimer();

            kernEfficientScan <<< fullBlockPerGrid, blockSize, 2 * N * sizeof(int) >>> (N, dev_out, dev_in);
            checkCUDAError("kernNaiveScan dev_in failed");


            timer().endGpuTimer();

            hipMemcpy(odata, dev_out, n * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy DeviceToHost failed");

            hipFree(dev_in);
            hipFree(dev_out);

        }
    }
}
