#include "hip/hip_runtime.h"
#include "common.h"


namespace StreamCompaction {
    namespace Common {

        /**
         * Maps an array to an array of 0s and 1s for stream compaction. Elements
         * which map to 0 will be removed, and elements which map to 1 will be kept.
         */
        __global__ void kernMapToBoolean(int n, int *bools, const PathSegment *idata) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index < n){
                bools[index] = (idata[index].remainingBounces != 0);
            }
        }

        /**
         * Performs scatter on an array. That is, for each element in idata,
         * if bools[idx] == 1, it copies idata[idx] to odata[indices[idx]].
         */
        __global__ void kernScatter(int n, PathSegment *odata,
                const PathSegment *idata, const int *bools, const int *indices) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index < n && bools[index]){
                odata[indices[index]] = idata[index];
            }
        }

    }
}
