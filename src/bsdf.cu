#include <hip/hip_runtime.h>
#include "glm/glm.hpp"

#include "sceneStructs.h"
#include "warpfunctions.cu"

enum BxDFType
{
	BSDF_REFLECTION = 1 << 0,   // This BxDF handles rays that are reflected off surfaces
	BSDF_TRANSMISSION = 1 << 1, // This BxDF handles rays that are transmitted through surfaces
	BSDF_DIFFUSE = 1 << 2,      // This BxDF represents diffuse energy scattering, which is uniformly random
	BSDF_GLOSSY = 1 << 3,       // This BxDF represents glossy energy scattering, which is biased toward certain directions
	BSDF_SPECULAR = 1 << 4,     // This BxDF handles specular energy scattering, which has no element of randomness
	BSDF_ALL = BSDF_DIFFUSE | BSDF_GLOSSY | BSDF_SPECULAR | BSDF_REFLECTION | BSDF_TRANSMISSION
};

namespace BSDF
{
	namespace
	{
		// Lamberts Material-------------------------------------------------------
		__host__ __device__ glm::vec3 Lamberts_F(const glm::vec3* wo, const glm::vec3* wi, const Material* material)
		{
			return material->color * Common::InvPi;
		}

		__host__ __device__ float Lamberts_Pdf(const glm::vec3* wo, glm::vec3* wi)
		{
			return Common::SameHemisphere(wo, wi) ? Common::AbsCosTheta(wi) * Common::InvPi : 0;
		}

		__host__ __device__ glm::vec3 Lamberts_sampleF(const glm::vec3* wo, glm::vec3* wi, float* pdf, const glm::vec2* xi, const Material* material)
		{
			// 1. Cosine sample the hemisphere
			*wi = WarpFunctions::SquareToHemisphereCosine(xi);

			if (wo->z < 0) {
				wi->z = -wi->z;
			}

			// 2. Calculate the pdf
			*pdf = Lamberts_Pdf(wo, wi);

			// 3. return f
			return Lamberts_F(wo, wi, material);
		}

		// TODO: Add more bxdfs

	} // Anonymous namespace end

	__host__ __device__ glm::vec3 Sample_F(const glm::vec3* woW, glm::vec3* wiW, float* pdf, glm::vec2* xi, const Material* material, const ShadeableIntersection* intersection)
	{
		// 1. Select Random Bxdf

		// 2. Rewriting the random number

		// 3. Converting wo, wi to tangent space

		// 4. Getting the color of the random bxdf
		glm::vec3 color(0.f);

		// if it is glass material, then we dont need to check other bxdf as it will only reflect in one direction

		// 5. Finding the average pdf of the remaining bxdfs

		// 6. Iterate through bxdf and sum result of f()

		return color;
	}

	__host__ __device__ glm::vec3 F(const glm::vec3* woW, const glm::vec3* wiW, const BxDFType flags)
	{
		glm::vec3 color(0.f);

		return color;
	}
	
	__host__ __device__ float Pdf(const glm::vec3* woW, const glm::vec3* wiw, const BxDFType flags)
	{
		return 0.f;
	}







} // namespace BSDF end







