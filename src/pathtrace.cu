#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/partition.h>
#include <thrust/device_ptr.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"
#include "warpfunctions.h"
#include "materials.h"
#include "lights.h"

#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)

void checkCUDAErrorFn(const char* msg, const char* file, int line)
{
#if ERRORCHECK
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (hipSuccess == err)
  {
    return;
  }

  fprintf(stderr, "CUDA error");
  if (file)
  {
    fprintf(stderr, " (%s:%d)", file, line);
  }
  fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
  getchar();
#  endif
  exit(EXIT_FAILURE);
#endif
}

__host__ __device__

thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth)
{
  int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
  return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
                               int iter, glm::vec3* image)
{
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;

  if (x < resolution.x && y < resolution.y)
  {
    int index = x + (y * resolution.x);
    glm::vec3 pix = image[index];

    glm::ivec3 color;
    color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
    color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
    color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);

    // Each thread writes one pixel location in the texture (textel)
    pbo[index].w = 0;
    pbo[index].x = color.x;
    pbo[index].y = color.y;
    pbo[index].z = color.z;
  }
}

static Scene* hst_scene = NULL;
static glm::vec3* dev_image = NULL;
static Geom* dev_geoms = NULL;
static Geom* dev_geom_lights = NULL;
static Material* dev_materials = NULL;
static PathSegment* dev_paths = NULL;
static ShadeableIntersection* dev_intersections = NULL;
static thrust::device_ptr<PathSegment> dev_thrust_paths;
// TODO: static variables for device memory, any extra info you need, etc
// ...

void pathtraceInit(Scene* scene)
{
  hst_scene = scene;
  const Camera& cam = hst_scene->state.camera;
  const int pixelcount = cam.resolution.x * cam.resolution.y;

  hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
  hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

  hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

  dev_thrust_paths = thrust::device_ptr<PathSegment>(dev_paths);

  hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
  hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);
  
  std::vector<Geom> lights;
  for(const auto& geom : scene->geoms)
  {
    if (scene->materials[geom.materialid].emittance <= 0)
    {
      continue;
    }

    lights.push_back(geom);
  }

  scene->m_numLights = lights.size();

  hipMalloc(&dev_geom_lights, lights.size() * sizeof(Geom));
  hipMemcpy(dev_geom_lights, lights.data(), lights.size() * sizeof(Geom), hipMemcpyHostToDevice);

  hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
  hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material),
             hipMemcpyHostToDevice);

  hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
  hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

  checkCUDAError("pathtraceInit");
}

void pathtraceFree()
{
  hipFree(dev_image); // no-op if dev_image is null
  hipFree(dev_paths);
  hipFree(dev_geoms);
  hipFree(dev_materials);
  hipFree(dev_intersections);
  // TODO: clean up any extra device memory you created

  checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;

  if (x < cam.resolution.x && y < cam.resolution.y)
  {
    int index = x + (y * cam.resolution.x);
    PathSegment& segment = pathSegments[index];

    segment.ray.origin = cam.position;
    segment.color = glm::vec3(0.0f, 0.0f, 0.0f);
    segment.throughput = glm::vec3(1.0f, 1.0f, 1.0f);

    // TODO: implement antialiasing by jittering the ray
    segment.ray.direction = glm::normalize(cam.view
      - cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f)
      - cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f)
    );

    segment.pixelIndex = index;
    segment.remainingBounces = traceDepth;
  }
}

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
  int depth
  , int num_paths
  , PathSegment* pathSegments
  , Geom* geoms
  , int geoms_size
  , ShadeableIntersection* intersections
)
{
  int path_index = blockIdx.x * blockDim.x + threadIdx.x;

  if (path_index < num_paths)
  {
    const PathSegment& pathSegment = pathSegments[path_index];
    int pixelIndex = pathSegment.pixelIndex;
    intersections[path_index] = Intersections::Do(pathSegment.ray, geoms, geoms_size);
  }
}

__device__ float PowerHeuristic(int nf, Float fPdf, int ng, Float gPdf)
{
  Float f = nf * fPdf;
  Float g = ng * gPdf;
  return (f * f) / (f * f + g * g);
}



// LOOK: "fake" shader demonstrating what you might do with the info in
// a ShadeableIntersection, as well as how to use thrust's random number
// generator. Observe that since the thrust random number generator basically
// adds "noise" to the iteration, the image should start off noisy and get
// cleaner as more iterations are computed.
//
// Note that this shader does NOT do a BSDF evaluation!
// Your shaders should handle that - this can allow techniques such as
// bump mapping.
__global__ void shadeRays(
  int iter,
  int maxDepth,
  int num_paths,
  int num_lights,
  int geoms_size,
  ShadeableIntersection* shadeableIntersections,
  PathSegment* pathSegments,
  Material* materials,
  Geom* lights,
  Geom* geoms
)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= num_paths)
  {
    return;
  }

  PathSegment& targetSegment = pathSegments[idx];
  const int pixelIndex = targetSegment.pixelIndex;

  // Didn't hit anything or hit something behind
  const ShadeableIntersection intersection = shadeableIntersections[idx];
  if (intersection.t <= 0.0f)
  {
    targetSegment.remainingBounces = 0;
    return;
  }

  // if the intersection exists...
  // Set up the RNG
  // LOOK: this is how you use thrust's RNG! Please look at
  // makeSeededRandomEngine as well.
  thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);
  thrust::uniform_real_distribution<float> u01(0, 1);

  const Material material = materials[intersection.materialId];

  // If the material indicates that the object was a light, "light" the ray
  if (material.emittance > 0.0f)
  {
    // Max Depth - Hit Light Directly
    if (maxDepth == pathSegments[idx].remainingBounces)
    {
      targetSegment.color += (targetSegment.throughput * material.color * material.emittance);
    }

    // Terminate Ray
    targetSegment.remainingBounces = 0;
    return;
  }

  const glm::vec3 woW = -targetSegment.ray.direction;
  const glm::vec3 wo = intersection.worldToTangent * woW;
  glm::vec3 directWiW;
  float directPdf = 0.0f;

  glm::vec3 indirectWi;
  glm::vec3 indirectWiW = glm::vec3(pixelIndex);
  float indirectPdf = 0.0f;
  Color3f indirectFrTerm;

  Color3f finalColor = Color3f(0.0f);

  const int randomIdx = (int)(u01(rng) * num_lights);
  Geom* activeLight = &lights[randomIdx];
  const Material lightMaterial = materials[activeLight->materialid];

  Intersection lightIntr;

  float directFactor = 0.0f;
  float indirectFactor = 0.0f;

  if (material.type == DIFFUSE)
  {
    indirectFrTerm = BRDF::Lambert::Sample_f(material.color, wo, &indirectWi, &indirectPdf, u01(rng), u01(rng));
  }

  indirectWiW = intersection.tangentToWorld * indirectWi;

  const Color3f directLi = Lights::Arealight::Sample_Li(lightMaterial.color * lightMaterial.emittance, intersection.intersectPoint, u01(rng), u01(rng), activeLight, &directWiW, &directPdf, &lightIntr);
  directPdf = directPdf / static_cast<float>(num_lights);

  if (directPdf > EPSILON)
  {
    const Ray shadowRay = Intersections::SpawnRay(intersection.intersectPoint, intersection.surfaceNormal, directWiW);
    const auto shadowIntr = Intersections::Do(shadowRay, geoms, geoms_size);
    
    if (shadowIntr.geom != nullptr)
    {
      // ID compare
      if (shadowIntr.geom->id == activeLight->id)
      {
        const float directCosTerm = std::abs(glm::dot(intersection.surfaceNormal, directWiW));
        const glm::vec3 directWi = intersection.worldToTangent * directWiW;

        if (material.type == DIFFUSE)
        {
          const Color3f directFrTerm = BRDF::Lambert::f(material.color, wo, directWi);
          directFactor = PowerHeuristic(1, directPdf, 1, BRDF::Lambert::Pdf(wo, directWi));
          finalColor += ((directFrTerm * directLi * directCosTerm * directFactor) / directPdf);
        }
      }
    }
  }

  if (indirectPdf > EPSILON)
  {
    float lightPdf = Lights::Arealight::Pdf_Li(intersection.intersectPoint, intersection.surfaceNormal, indirectWiW, activeLight);
    if (lightPdf > EPSILON) {
      lightPdf = lightPdf / num_lights;
      indirectFactor = PowerHeuristic(1, indirectPdf, 1, lightPdf);
    }
  
    Ray indirectRay = Intersections::SpawnRay(intersection.intersectPoint, intersection.surfaceNormal, indirectWiW);
    Intersection indirectIsect;
  
    const float indirectCosTerm = std::abs(glm::dot(intersection.surfaceNormal, indirectWiW));
  
    const auto indirectIntr = Intersections::Do(indirectRay, geoms, geoms_size);
  
    Color3f indirectLiTerm = Color3f(0.0f);
  
    if (indirectIntr.geom != nullptr)
    {
      if (indirectIntr.geom->id == activeLight->id) {
        indirectLiTerm = Lights::Arealight::L(lightMaterial.color * lightMaterial.emittance, indirectIntr.surfaceNormal, -indirectWiW);
      }
  
      finalColor += ((indirectFrTerm * indirectLiTerm * indirectCosTerm * indirectFactor)  / indirectPdf);
    }
  }

  // Add MIS Color
  targetSegment.color += (finalColor * targetSegment.throughput);

  targetSegment.remainingBounces--;

  if (targetSegment.remainingBounces <= 0)
  {
    // No Need to compute next ray
    return;
  }

  Vector3f bounceWi;
  Vector3f bounceWiW;
  float bouncePdf;
  Color3f bounceFrTerm;

  if (material.type == DIFFUSE)
  {
    bounceFrTerm = BRDF::Lambert::Sample_f(material.color, wo, &bounceWi, &bouncePdf, u01(rng), u01(rng));
  }

  bounceWiW = intersection.tangentToWorld * bounceWi;

  const float bounceCosTerm = std::abs(glm::dot(intersection.surfaceNormal, bounceWiW));

  if (bouncePdf < EPSILON) {
    // Terminate Ray
    targetSegment.remainingBounces = 0;
    return;
  }

  targetSegment.throughput = (targetSegment.throughput * bounceFrTerm * bounceCosTerm) / bouncePdf;
  targetSegment.ray = Intersections::SpawnRay(intersection.intersectPoint, intersection.surfaceNormal, bounceWiW);

  // Russian Roulette
  const float maxVal = glm::max(glm::max(static_cast<float>(targetSegment.throughput[0]), targetSegment.throughput[1]), targetSegment.throughput[2]);

  if (u01(rng) < (1.0f - maxVal)) {
    targetSegment.remainingBounces = 0;
    return;
  }
  
  targetSegment.throughput /= maxVal;
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3* image, PathSegment* iterationPaths)
{
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (index < nPaths)
  {
    PathSegment iterationPath = iterationPaths[index];
    image[iterationPath.pixelIndex] += iterationPath.color;
  }
}

struct IsValidPath
{
  __host__ __device__ bool operator() (const PathSegment& segment)
  {
    return segment.remainingBounces > 0;
  }
};

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4* pbo, int frame, int iter)
{
  const int traceDepth = hst_scene->state.traceDepth;
  const Camera& cam = hst_scene->state.camera;
  const int pixelcount = cam.resolution.x * cam.resolution.y;

  // 2D block for generating ray from camera
  const dim3 blockSize2d(8, 8);
  const dim3 blocksPerGrid2d(
    (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
    (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

  // 1D block for path tracing
  const int blockSize1d = 256;

  ///////////////////////////////////////////////////////////////////////////

  // Recap:
  // * Initialize array of path rays (using rays that come out of the camera)
  //   * You can pass the Camera object to that kernel.
  //   * Each path ray must carry at minimum a (ray, color) pair,
  //   * where color starts as the multiplicative identity, white = (1, 1, 1).
  //   * This has already been done for you.
  // * For each depth:
  //   * Compute an intersection in the scene for each path ray.
  //     A very naive version of this has been implemented for you, but feel
  //     free to add more primitives and/or a better algorithm.
  //     Currently, intersection distance is recorded as a parametric distance,
  //     t, or a "distance along the ray." t = -1.0 indicates no intersection.
  //     * Color is attenuated (multiplied) by reflections off of any object
  //   * TODO: Stream compact away all of the terminated paths.
  //     You may use either your implementation or `thrust::remove_if` or its
  //     cousins.
  //     * Note that you can't really use a 2D kernel launch any more - switch
  //       to 1D.
  //   * TODO: Shade the rays that intersected something or didn't bottom out.
  //     That is, color the ray by performing a color computation according
  //     to the shader, then generate a new ray to continue the ray path.
  //     We recommend just updating the ray's PathSegment in place.
  //     Note that this step may come before or after stream compaction,
  //     since some shaders you write may also cause a path to terminate.
  // * Finally, add this iteration's results to the image. This has been done
  //   for you.

  // TODO: perform one iteration of path tracing

  generateRayFromCamera <<<blocksPerGrid2d, blockSize2d >>>(cam, iter, traceDepth, dev_paths);
  checkCUDAError("generate camera ray");

  int depth = 0;
  PathSegment* dev_path_end = dev_paths + pixelcount;
  int num_paths = dev_path_end - dev_paths;

  const int all_path_count = num_paths;

  // --- PathSegment Tracing Stage ---
  // Shoot ray into scene, bounce between objects, push shading chunks

  bool iterationComplete = false;
  while (!iterationComplete)
  {
    // clean shading chunks
    hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    // tracing
    dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
    computeIntersections <<<numblocksPathSegmentTracing, blockSize1d>>>(
      depth,
      num_paths,
      dev_paths,
      dev_geoms,
      hst_scene->geoms.size(),
      dev_intersections
    );
    checkCUDAError("trace one bounce");
    hipDeviceSynchronize();
    depth++;

    shadeRays<<<numblocksPathSegmentTracing, blockSize1d>>>(
      iter,
      traceDepth,
      num_paths,
      hst_scene->m_numLights,
      hst_scene->geoms.size(),
      dev_intersections,
      dev_paths,
      dev_materials,
      dev_geom_lights,
      dev_geoms
    );

    const auto middleItr = thrust::partition(dev_thrust_paths, dev_thrust_paths + num_paths, IsValidPath());
    iterationComplete = dev_paths == middleItr.get();
    num_paths = middleItr.get() - dev_paths;
  }

  hipDeviceSynchronize();

  // Assemble this iteration and apply it to the image
  dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
  finalGather<<<numBlocksPixels, blockSize1d>>>(all_path_count, dev_image, dev_paths);

  ///////////////////////////////////////////////////////////////////////////

  // Send results to OpenGL buffer for rendering
  sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);

  // Retrieve image from GPU
  hipMemcpy(hst_scene->state.image.data(), dev_image,
             pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

  checkCUDAError("pathtrace");
}
