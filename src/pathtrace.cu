#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/partition.h>
#include <thrust/device_ptr.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"
#include "warpfunctions.h"
#include "materials.h"
#include "lights.h"

// #define SORT_INTERSECTIONS_BY_MATERIAL_ID

// Enable Either One:
#define FULL_LIGHTING_INTEGRATOR
// #define DIRECT_LIGHTING_INTEGRATOR

// #define DEBUG_NORMALS
// #define DEBUG_TANGENTS
// #define DEBUG_BITANGENTS
// #define DEBUG_UV
// #define DEBUG_ROUGHNESS
// #define DEBUG_DIFFUSE

#define USE_NORMAL_MAPS
#define USE_DIFFUSE_MAPS
// #define USE_EMISSIVE_MAPS
#define USE_ROUGHNESS_MAPS

#define USE_PLASTIC_MATERIAL

#define ENABLE_ANTI_ALIASING

#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)

void checkCUDAErrorFn(const char* msg, const char* file, int line)
{
#if ERRORCHECK
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (hipSuccess == err)
  {
    return;
  }

  fprintf(stderr, "CUDA error");
  if (file)
  {
    fprintf(stderr, " (%s:%d)", file, line);
  }
  fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
  getchar();
#  endif
  exit(EXIT_FAILURE);
#endif
}

__host__ __device__

thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth)
{
  int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
  return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
                               int iter, glm::vec3* image)
{
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;

  if (x < resolution.x && y < resolution.y)
  {
    int index = x + (y * resolution.x);
    glm::vec3 pix = image[index];

    glm::ivec3 color;
    color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
    color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
    color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);

    // Each thread writes one pixel location in the texture (textel)
    pbo[index].w = 0;
    pbo[index].x = color.x;
    pbo[index].y = color.y;
    pbo[index].z = color.z;
  }
}

static Scene* hst_scene = NULL;
static glm::vec3* dev_image = NULL;
static Geom* dev_geoms = NULL;
static Geom* dev_geom_lights = NULL;
static Material* dev_materials = NULL;
static int* dev_path_material_ids = NULL;
static int* dev_path_indices = NULL;
static Triangle* dev_triangles = NULL;
static PathSegment* dev_paths = NULL;
static PathSegment* dev_paths_b = NULL;
static ImageInfo* dev_imageInfo = NULL;
static glm::vec3* dev_texels = NULL;
static KDNode* dev_kd_nodes = NULL;
static Triangle* dev_kd_triangles = NULL;
thrust::device_ptr<PathSegment> dev_thrust_paths;
static ShadeableIntersection* dev_intersections = NULL;
static ShadeableIntersection* dev_intersections_b = NULL;
static thrust::device_ptr<int> dev_thrust_path_indices;
static thrust::device_ptr<int> dev_thrust_path_material_ids;
// TODO: static variables for device memory, any extra info you need, etc
// ...

void pathtraceInit(Scene* scene)
{
  hst_scene = scene;
  const Camera& cam = hst_scene->state.camera;
  const int pixelcount = cam.resolution.x * cam.resolution.y;

  hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
  hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

  hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));
  hipMalloc(&dev_paths_b, pixelcount * sizeof(PathSegment));
  hipMalloc(&dev_path_material_ids, pixelcount * sizeof(int));
  hipMalloc(&dev_path_indices, pixelcount * sizeof(int));

  dev_thrust_path_material_ids = thrust::device_ptr<int>(dev_path_material_ids);
  dev_thrust_path_indices = thrust::device_ptr<int>(dev_path_indices);

  hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
  hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);
  
  hipMalloc(&dev_imageInfo, scene->imageInfo.size() * sizeof(ImageInfo));
  hipMemcpy(dev_imageInfo, scene->imageInfo.data(), scene->imageInfo.size() * sizeof(ImageInfo), hipMemcpyHostToDevice);

  hipMalloc(&dev_texels, scene->allTexels.size() * sizeof(glm::vec3));
  hipMemcpy(dev_texels, scene->allTexels.data(), scene->allTexels.size() * sizeof(glm::vec3), hipMemcpyHostToDevice);

  hipMalloc(&dev_triangles, scene->meshTriangles.size() * sizeof(Triangle));
  hipMemcpy(dev_triangles, scene->meshTriangles.data(), scene->meshTriangles.size() * sizeof(Triangle), hipMemcpyHostToDevice);

  hipMalloc(&dev_kd_nodes, scene->nodes.size() * sizeof(KDNode));
  hipMemcpy(dev_kd_nodes, scene->nodes.data(), scene->nodes.size() * sizeof(KDNode), hipMemcpyHostToDevice);

  hipMalloc(&dev_kd_triangles, scene->nodeTriangles.size() * sizeof(Triangle));
  hipMemcpy(dev_kd_triangles, scene->nodeTriangles.data(), scene->nodeTriangles.size() * sizeof(Triangle), hipMemcpyHostToDevice);

  std::vector<Geom> lights;
  for(const auto& geom : scene->geoms)
  {
    if (scene->materials[geom.materialid].emittance <= 0)
    {
      continue;
    }

    lights.push_back(geom);
  }

  scene->m_numLights = int(lights.size());

  hipMalloc(&dev_geom_lights, lights.size() * sizeof(Geom));
  hipMemcpy(dev_geom_lights, lights.data(), lights.size() * sizeof(Geom), hipMemcpyHostToDevice);

  hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
  hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material),
             hipMemcpyHostToDevice);

  hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
  hipMalloc(&dev_intersections_b, pixelcount * sizeof(ShadeableIntersection));
  hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));
  hipMemset(dev_intersections_b, 0, pixelcount * sizeof(ShadeableIntersection));

  checkCUDAError("pathtraceInit");
}

void pathtraceFree()
{
  hipFree(dev_image); // no-op if dev_image is null
  hipFree(dev_paths);
  hipFree(dev_geoms);
  hipFree(dev_materials);
  hipFree(dev_intersections);
  hipFree(dev_geom_lights);
  hipFree(dev_path_material_ids);
  hipFree(dev_path_indices);
  hipFree(dev_paths_b);
  hipFree(dev_imageInfo);
  hipFree(dev_texels);
  hipFree(dev_triangles);
  hipFree(dev_intersections_b);
  hipFree(dev_kd_nodes);
  hipFree(dev_kd_triangles);

  checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;

  if (x < cam.resolution.x && y < cam.resolution.y)
  {
    int index = x + (y * cam.resolution.x);
    PathSegment& segment = pathSegments[index];

    segment.ray.origin = cam.position;
    segment.color = glm::vec3(0.0f, 0.0f, 0.0f);
    segment.throughput = glm::vec3(1.0f, 1.0f, 1.0f);

#ifdef ENABLE_ANTI_ALIASING
    thrust::default_random_engine rng = makeSeededRandomEngine(iter, 37, 0);
    thrust::uniform_real_distribution<float> u01(0, 1);

    segment.ray.direction = glm::normalize(cam.view
      - cam.right * cam.pixelLength.x * ((float)(x + u01(rng)) - (float)cam.resolution.x * 0.5f)
      - cam.up * cam.pixelLength.y * ((float)(y + u01(rng)) - (float)cam.resolution.y * 0.5f)
    );
#else
    segment.ray.direction = glm::normalize(cam.view
      - cam.right * cam.pixelLength.x * ((float)(x) - (float)cam.resolution.x * 0.5f)
      - cam.up * cam.pixelLength.y * ((float)(y) - (float)cam.resolution.y * 0.5f)
    );
#endif

    segment.pixelIndex = index;
    segment.remainingBounces = traceDepth;
    segment.rayFromSpecular = false;
  }
}

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
  int depth
  , int num_paths
  , PathSegment* pathSegments
  , Geom* geoms
  , int geoms_size
  , ShadeableIntersection* intersections,
  Triangle* triangles
  , KDNode* kdNodes, Triangle* kdTriangles
)
{
  int path_index = blockIdx.x * blockDim.x + threadIdx.x;

  if (path_index < num_paths)
  {
    const PathSegment& pathSegment = pathSegments[path_index];
    intersections[path_index] = Intersections::Do(pathSegment.ray, geoms, geoms_size, triangles, kdNodes, kdTriangles);
  }
}

__device__ float PowerHeuristic(int nf, Float fPdf, int ng, Float gPdf)
{
  Float f = nf * fPdf;
  Float g = ng * gPdf;
  return (f * f) / (f * f + g * g);
}


__device__ Color3f GetTextureData(const ImageInfo& info, const glm::vec2& targetUV, glm::vec3* texels)
{
  const glm::vec2 uv (glm::fract(targetUV.y * info.repeatX), glm::fract(targetUV.x * info.repeatY));

  const int X = glm::min(info.width * uv.x, info.width - 1.0f);
  const int Y = glm::min(info.height * uv.y, info.height - 1.0f); // flipped from stb image
  const int linearCoordinate = info.startIdx + X + info.height * Y;
  return texels[linearCoordinate];
}

// LOOK: "fake" shader demonstrating what you might do with the info in
// a ShadeableIntersection, as well as how to use thrust's random number
// generator. Observe that since the thrust random number generator basically
// adds "noise" to the iteration, the image should start off noisy and get
// cleaner as more iterations are computed.
//
// Note that this shader does NOT do a BSDF evaluation!
// Your shaders should handle that - this can allow techniques such as
// bump mapping.
__global__ void shadeRays(
  int iter,
  int maxDepth,
  int num_paths,
  int num_lights,
  int geoms_size,
  ShadeableIntersection* shadeableIntersections,
  PathSegment* pathSegments,
  Material* materials,
  Geom* lights,
  Geom* geoms,
  glm::vec3* texels,
  ImageInfo* imageInfos,
  Triangle* triangles
  , KDNode* kdNodes, Triangle* kdTriangles
)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= num_paths)
  {
    return;
  }

  PathSegment& targetSegment = pathSegments[idx];

  // Didn't hit anything or hit something behind
  ShadeableIntersection intersection = shadeableIntersections[idx];
  if (intersection.t <= 0.0f)
  {
    targetSegment.remainingBounces = 0;
    return;
  }

  // if the intersection exists...
  // Set up the RNG
  // LOOK: this is how you use thrust's RNG! Please look at
  // makeSeededRandomEngine as well.
  thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);
  thrust::uniform_real_distribution<float> u01(0, 1);

  const Material material = materials[intersection.materialId];
  bool sampledSpecular = material.type == SPECULAR || material.type == ROUGH_SPECULAR || material.type == TRANSMISSIVE || material.type == GLASS;

  // If the material indicates that the object was a light, "light" the ray
  if (material.emittance > 0.0f)
  {
    // Max Depth - Hit Light Directly
    if (maxDepth == pathSegments[idx].remainingBounces || targetSegment.rayFromSpecular)
    {
      targetSegment.color += (targetSegment.throughput * material.color * material.emittance);
    }

    // Terminate Ray
    targetSegment.remainingBounces = 0;
    return;
  }

#ifdef USE_NORMAL_MAPS
  if (material.normalMapId >= 0)
  {
    const Normal3f normalColor = GetTextureData(imageInfos[material.normalMapId], intersection.uv, texels);
    const Normal3f normalValue = glm::normalize(normalColor * 2.0f - glm::vec3(1.0f));

    Vector3f tangent;
    Vector3f bitangent;

    intersection.surfaceNormal = intersection.tangentToWorld * normalValue;
    CoordinateSystem(intersection.surfaceNormal, &tangent, &bitangent);

    intersection.tangentToWorld = glm::mat3(
      tangent,
      bitangent,
      intersection.surfaceNormal
    );

    intersection.worldToTangent = glm::transpose(intersection.tangentToWorld);
  }
#endif

#ifdef DEBUG_NORMALS
  targetSegment.color = (intersection.surfaceNormal + Vector3f(1, 1, 1)) / 2.0f;
  targetSegment.remainingBounces = 0;
  return;
#endif
  
#ifdef DEBUG_TANGENTS
  targetSegment.color = (intersection.surfaceTangent + Vector3f(1, 1, 1)) / 2.0f;
  targetSegment.remainingBounces = 0;
  return;
#endif
  
#ifdef DEBUG_BITANGENTS
  targetSegment.color = (intersection.surfaceBitangent + Vector3f(1, 1, 1)) / 2.0f;
  targetSegment.remainingBounces = 0;
  return;
#endif

#ifdef DEBUG_UV
  targetSegment.color = Vector3f(intersection.uv.x, intersection.uv.y, 0);
  targetSegment.remainingBounces = 0;
  return;
#endif

  const glm::vec3 woW = -targetSegment.ray.direction;
  const glm::vec3 wo = intersection.worldToTangent * woW;
  glm::vec3 WiW;
  float pdf = 0.0f;

  Color3f finalColor = Color3f(0.0f);

  const int randomIdx = static_cast<int>(u01(rng) * num_lights);
  Geom* activeLight = &lights[randomIdx];
  const Material lightMaterial = materials[activeLight->materialid];

  Color3f diffuseMaterialColor = material.color;
  float materialRoughness = material.roughness;
#ifdef USE_DIFFUSE_MAPS
  if (material.diffuseMapId >= 0)
  {
    diffuseMaterialColor = GetTextureData(imageInfos[material.diffuseMapId], intersection.uv, texels);
  }
#endif

#ifdef DEBUG_DIFFUSE
  targetSegment.color = diffuseMaterialColor;
  targetSegment.remainingBounces = 0;
  return;
#endif

#ifdef USE_ROUGHNESS_MAPS
  if (material.roughMapId >= 0)
  {
    const Color3f roughnessColor = GetTextureData(imageInfos[material.roughMapId], intersection.uv, texels);
    materialRoughness = roughnessColor.r / 255.0f;
  }
#endif

#ifdef DEBUG_ROUGHNESS
  targetSegment.color = Vector3f(materialRoughness * 255.0f);
  targetSegment.remainingBounces = 0;
  return;
#endif

#ifdef USE_PLASTIC_MATERIAL
  if (material.type == PLASTIC)
  {
    const float bxdfSelect = u01(rng);
    sampledSpecular = bxdfSelect < material.hasReflective;
  }
#endif

  if (!sampledSpecular) {
    const Color3f directLi = Lights::Arealight::Sample_Li(lightMaterial.color * lightMaterial.emittance, intersection.intersectPoint, intersection.surfaceNormal, u01(rng), u01(rng), activeLight, &WiW, &pdf);
    pdf = pdf / static_cast<float>(num_lights);

    if (pdf > EPSILON)
    {
      const Ray shadowRay = Intersections::SpawnRay(intersection.intersectPoint, intersection.surfaceNormal, WiW);
      const auto shadowIntr = Intersections::Do(shadowRay, geoms, geoms_size, triangles, kdNodes, kdTriangles);

      if (shadowIntr.geom != nullptr)
      {
        // ID compare
        if (shadowIntr.geom->id == activeLight->id)
        {
          const float directCosTerm = std::abs(glm::dot(intersection.surfaceNormal, WiW));
          const glm::vec3 directWi = intersection.worldToTangent * WiW;

          if (material.type == DIFFUSE)
          {
            const Color3f directFrTerm = BRDF::Lambert::f(diffuseMaterialColor, wo, directWi, materialRoughness);
            const float directFactor = PowerHeuristic(1, pdf, 1, BRDF::Lambert::Pdf(wo, directWi));
            finalColor += ((directFrTerm * directLi * directCosTerm * directFactor) / pdf);
          }
#ifdef USE_PLASTIC_MATERIAL
          else if (material.type == PLASTIC)
          {
            const bool reflect = glm::dot(WiW, intersection.surfaceNormal) * glm::dot(WiW, intersection.surfaceNormal) > 0;
            Color3f directFrTerm = Color3f(0.0f);

            if (reflect)
            {
              directFrTerm += BRDF::Microfacet::f(material.ks, wo, directWi, FRESNEL_DIELECTRIC, materialRoughness, materialRoughness, Color3f(1.5f), Color3f(1.5f));
            }

            directFrTerm += BRDF::Lambert::f(diffuseMaterialColor, wo, directWi, materialRoughness);

            const float directFactor = PowerHeuristic(1, pdf, 1, BRDF::Lambert::Pdf(wo, directWi));
            finalColor += ((directFrTerm * directLi * directCosTerm * directFactor) / pdf);
          }
#endif
        }
      }
    }

    const Color3f indirectFrTerm = BRDF::Lambert::Sample_f(diffuseMaterialColor, wo, &WiW, &pdf, u01(rng), u01(rng), materialRoughness);

#ifdef USE_PLASTIC_MATERIAL
    if(material.type == PLASTIC)
    {
      pdf = pdf / 2.0f;
    }
#endif

    WiW = intersection.tangentToWorld * WiW;

    if (pdf > EPSILON)
    {
      float indirectFactor = 0.0f;

      float lightPdf = Lights::Arealight::Pdf_Li(intersection.intersectPoint, intersection.surfaceNormal, WiW, activeLight);
      if (lightPdf > EPSILON) {
        lightPdf = lightPdf / num_lights;
        indirectFactor = PowerHeuristic(1, pdf, 1, lightPdf);
      }

      const Ray indirectRay = Intersections::SpawnRay(intersection.intersectPoint, intersection.surfaceNormal, WiW);

      const float indirectCosTerm = std::abs(glm::dot(intersection.surfaceNormal, WiW));

      const auto indirectIntr = Intersections::Do(indirectRay, geoms, geoms_size, triangles, kdNodes, kdTriangles);

      Color3f indirectLiTerm = Color3f(0.0f);

      if (indirectIntr.geom != nullptr)
      {
        if (indirectIntr.geom->id == activeLight->id) {
          indirectLiTerm = Lights::Arealight::L(lightMaterial.color * lightMaterial.emittance, indirectIntr.surfaceNormal, -WiW);
        }

        finalColor += ((indirectFrTerm * indirectLiTerm * indirectCosTerm * indirectFactor) / pdf);
      }
    }
  }

  if (material.emissiveMapId >= 0)
  {
    finalColor += GetTextureData(imageInfos[material.emissiveMapId], intersection.uv, texels);
  }

  // Add MIS Color
  targetSegment.color += (finalColor * targetSegment.throughput);

  targetSegment.remainingBounces--;

  if (targetSegment.remainingBounces <= 0)
  {
    // No Need to compute next ray
    return;
  }

  Color3f bounceFrTerm;

  targetSegment.rayFromSpecular = false;

  if (material.type == DIFFUSE)
  {
    bounceFrTerm = BRDF::Lambert::Sample_f(diffuseMaterialColor, wo, &WiW, &pdf, u01(rng), u01(rng), materialRoughness);
  }
  else if (material.type == SPECULAR)
  {
    bounceFrTerm = BRDF::Specular::Sample_f(diffuseMaterialColor, wo, &WiW, &pdf, FRESNEL_NOOP,  1.0f, material.indexOfRefraction);
    targetSegment.rayFromSpecular = true;
  }
  else if (material.type == ROUGH_SPECULAR)
  {
    bounceFrTerm = BRDF::Microfacet::Sample_f(material.color, wo, &WiW, &pdf, FRESNEL_NOOP, u01(rng), u01(rng), material.roughness, material.roughness, Color3f(1.0f), material.metalEta);
    targetSegment.rayFromSpecular = true;
  }
  else if (material.type == METAL)
  {
    bounceFrTerm = BRDF::Microfacet::Sample_f(material.color, wo, &WiW, &pdf, FRESNEL_CONDUCTOR, u01(rng), u01(rng), material.roughness, material.roughness,  Color3f(1.0f), material.metalEta);
    targetSegment.rayFromSpecular = true;
  }
  else if (material.type == TRANSMISSIVE)
  {
    bounceFrTerm = BRDF::SpecularBTDF::Sample_f(diffuseMaterialColor, wo, &WiW, &pdf, FRESNEL_NOREFLECT, 1.0f, material.indexOfRefraction);
    targetSegment.rayFromSpecular = true;
  }
  else if (material.type == GLASS)
  {
    const float bxdfSelect = u01(rng);

    if (bxdfSelect < material.hasReflective)
    {
      bounceFrTerm = BRDF::Specular::Sample_f(diffuseMaterialColor, wo, &WiW, &pdf, FRESNEL_NOOP, 1.0f, material.indexOfRefraction);
    }
    else
    {
      bounceFrTerm = BRDF::SpecularBTDF::Sample_f(material.kt, wo, &WiW, &pdf, FRESNEL_NOREFLECT, 1.0f, material.indexOfRefraction);
    }

    pdf = pdf / 2;

    targetSegment.rayFromSpecular = true;
  }
#ifdef USE_PLASTIC_MATERIAL
  else if (material.type == PLASTIC)
  {
    const float bxdfSelect = u01(rng);

    if (bxdfSelect < material.hasReflective)
    {
      bounceFrTerm = BRDF::Microfacet::Sample_f(material.color, wo, &WiW, &pdf, FRESNEL_DIELECTRIC, u01(rng), u01(rng), materialRoughness, materialRoughness, Color3f(1.0f), material.metalEta);
      targetSegment.rayFromSpecular = true;
    }
    else
    {
      bounceFrTerm = BRDF::Lambert::Sample_f(diffuseMaterialColor, wo, &WiW, &pdf, u01(rng), u01(rng), materialRoughness);
    }

    pdf = pdf / 2;
  }
#endif

  WiW = intersection.tangentToWorld * WiW;

  const float bounceCosTerm = std::abs(glm::dot(intersection.surfaceNormal, WiW));

  if (pdf < EPSILON) {
    // Terminate Ray
    targetSegment.remainingBounces = 0;
    return;
  }

  targetSegment.throughput = (targetSegment.throughput * bounceFrTerm * bounceCosTerm) / pdf;
  targetSegment.ray = Intersections::SpawnRay(intersection.intersectPoint, intersection.surfaceNormal, WiW);

  // Russian Roulette
  const float maxVal = glm::max(glm::max(static_cast<float>(targetSegment.throughput[0]), targetSegment.throughput[1]), targetSegment.throughput[2]);

  if (u01(rng) < (1.0f - maxVal)) {
    targetSegment.remainingBounces = 0;
    return;
  }
  
  targetSegment.throughput /= maxVal;
}

__global__ void shadeDirectLighting(
  int iter,
  int maxDepth,
  int num_paths,
  int num_lights,
  int geoms_size,
  ShadeableIntersection* shadeableIntersections,
  PathSegment* pathSegments,
  Material* materials,
  Geom* lights,
  Geom* geoms,
  glm::vec3* texels,
  ImageInfo* imageInfos,
  Triangle* triangles
  , KDNode* kdNodes, Triangle* kdTriangles
)
{
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= num_paths)
  {
    return;
  }

  PathSegment& targetSegment = pathSegments[idx];

  // Didn't hit anything or hit something behind
  ShadeableIntersection intersection = shadeableIntersections[idx];
  if (intersection.t <= 0.0f)
  {
    targetSegment.remainingBounces = 0;
    return;
  }

  thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);
  thrust::uniform_real_distribution<float> u01(0, 1);

  const Material material = materials[intersection.materialId];
  const bool sampledSpecular = material.type == SPECULAR || material.type == ROUGH_SPECULAR || material.type == METAL;

  // If the material indicates that the object was a light, "light" the ray
  if (material.emittance > 0.0f)
  {
    // Max Depth - Hit Light Directly
    if (maxDepth == pathSegments[idx].remainingBounces || targetSegment.rayFromSpecular)
    {
      targetSegment.color += (targetSegment.throughput * material.color * material.emittance);
    }

    // Terminate Ray
    targetSegment.remainingBounces = 0;
    return;
  }

#ifdef USE_NORMAL_MAPS
  if (material.normalMapId >= 0)
  {
    const Normal3f normalColor = GetTextureData(imageInfos[material.normalMapId], intersection.uv, texels);
    const Normal3f normalValue = glm::normalize(normalColor * 2.0f - glm::vec3(1.0f));

    intersection.surfaceNormal = intersection.tangentToWorld * normalValue;

    Vector3f tangent;
    Vector3f bitangent;

    CoordinateSystem(intersection.surfaceNormal, &tangent, &bitangent);

    intersection.tangentToWorld = glm::mat3(
      tangent,
      bitangent,
      intersection.surfaceNormal
    );

    intersection.worldToTangent = glm::transpose(intersection.tangentToWorld);
  }
#endif

  const glm::vec3 woW = -targetSegment.ray.direction;
  const glm::vec3 wo = intersection.worldToTangent * woW;

  Color3f finalColor = Color3f(0.0f);

  const int randomIdx = static_cast<int>(u01(rng) * num_lights);
  Geom* activeLight = &lights[randomIdx];
  const Material lightMaterial = materials[activeLight->materialid];

  Color3f diffuseMaterialColor = material.color;
  float materialRoughness = material.roughness;
#ifdef USE_DIFFUSE_MAPS
  if (material.diffuseMapId >= 0)
  {
    diffuseMaterialColor = GetTextureData(imageInfos[material.diffuseMapId], intersection.uv, texels);
  }
#endif

#ifdef DEBUG_DIFFUSE
  targetSegment.color = diffuseMaterialColor;
  targetSegment.remainingBounces = 0;
  return;
#endif

#ifdef USE_ROUGHNESS_MAPS
  if (material.roughMapId >= 0)
  {
    const Color3f roughnessColor = GetTextureData(imageInfos[material.roughMapId], intersection.uv, texels);
    materialRoughness = roughnessColor.r / 255.0f;
  }
#endif

#ifdef DEBUG_ROUGHNESS
  targetSegment.color = Vector3f(materialRoughness * 255.0f);
  targetSegment.remainingBounces = 0;
  return;
#endif

  if (!sampledSpecular) {
    glm::vec3 directWiW;
    float directPdf = 0.0f;

    const Color3f directLi = Lights::Arealight::Sample_Li(lightMaterial.color * lightMaterial.emittance, intersection.intersectPoint, intersection.surfaceNormal, u01(rng), u01(rng), activeLight, &directWiW, &directPdf);
    directPdf = directPdf / static_cast<float>(num_lights);

    if (directPdf > EPSILON)
    {
      const Ray shadowRay = Intersections::SpawnRay(intersection.intersectPoint, intersection.surfaceNormal, directWiW);
      const auto shadowIntr = Intersections::Do(shadowRay, geoms, geoms_size, triangles, kdNodes, kdTriangles);

      if (shadowIntr.geom != nullptr)
      {
        // ID compare
        if (shadowIntr.geom->id == activeLight->id)
        {
          const float directCosTerm = std::abs(glm::dot(intersection.surfaceNormal, directWiW));
          const Color3f directFrTerm = BRDF::Lambert::f(diffuseMaterialColor, wo, intersection.worldToTangent * directWiW, materialRoughness);
          finalColor += ((directFrTerm * directLi * directCosTerm) / directPdf);
        }
      }
    }
  }

#ifdef USE_EMISSIVE_MAPS
  if (material.emissiveMapId >= 0)
  {
    finalColor += GetTextureData(imageInfos[material.emissiveMapId], intersection.uv, texels);
  }
#endif

  // Add Direct Lighting Color & End
  targetSegment.color += (finalColor * targetSegment.throughput);
  targetSegment.remainingBounces = 0;
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3* image, PathSegment* iterationPaths)
{
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (index < nPaths)
  {
    PathSegment iterationPath = iterationPaths[index];
    image[iterationPath.pixelIndex] += iterationPath.color;
  }
}

__global__ void fillMaterialArray(int nPaths, int* materialIds, int* pathIndices, ShadeableIntersection* shadeableIntersections)
{
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (index < nPaths)
  {
    const ShadeableIntersection& shadeIntr = shadeableIntersections[index];
    materialIds[index] = shadeIntr.materialId;
    pathIndices[index] = index;
  }
}

__global__ void reshufflePathSegments(int nPaths, int* pathIndices, ShadeableIntersection* shadeableIntersections, PathSegment* segments,  ShadeableIntersection* oIntrs, PathSegment* oSegments)
{
  const int index = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (index < nPaths)
  {
    const int idx = pathIndices[index];
    oIntrs[index] = shadeableIntersections[idx];
    oSegments[index] = segments[idx];
  }
}


struct IsValidPath
{
  __host__ __device__ bool operator() (const PathSegment& segment)
  {
    return segment.remainingBounces > 0;
  }
};

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4* pbo, int frame, int iter)
{
  const int traceDepth = hst_scene->state.traceDepth;
  const Camera& cam = hst_scene->state.camera;
  const int pixelcount = cam.resolution.x * cam.resolution.y;

  // 2D block for generating ray from camera
  const dim3 blockSize2d(8, 8);
  const dim3 blocksPerGrid2d(
    (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
    (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

  // 1D block for path tracing
  const int blockSize1d = 256;

  ///////////////////////////////////////////////////////////////////////////

  // Recap:
  // * Initialize array of path rays (using rays that come out of the camera)
  //   * You can pass the Camera object to that kernel.
  //   * Each path ray must carry at minimum a (ray, color) pair,
  //   * where color starts as the multiplicative identity, white = (1, 1, 1).
  //   * This has already been done for you.
  // * For each depth:
  //   * Compute an intersection in the scene for each path ray.
  //     A very naive version of this has been implemented for you, but feel
  //     free to add more primitives and/or a better algorithm.
  //     Currently, intersection distance is recorded as a parametric distance,
  //     t, or a "distance along the ray." t = -1.0 indicates no intersection.
  //     * Color is attenuated (multiplied) by reflections off of any object
  //   * TODO: Stream compact away all of the terminated paths.
  //     You may use either your implementation or `thrust::remove_if` or its
  //     cousins.
  //     * Note that you can't really use a 2D kernel launch any more - switch
  //       to 1D.
  //   * TODO: Shade the rays that intersected something or didn't bottom out.
  //     That is, color the ray by performing a color computation according
  //     to the shader, then generate a new ray to continue the ray path.
  //     We recommend just updating the ray's PathSegment in place.
  //     Note that this step may come before or after stream compaction,
  //     since some shaders you write may also cause a path to terminate.
  // * Finally, add this iteration's results to the image. This has been done
  //   for you.

  // TODO: perform one iteration of path tracing

  generateRayFromCamera <<<blocksPerGrid2d, blockSize2d >>>(cam, iter, traceDepth, dev_paths);
  checkCUDAError("generate camera ray");

  int depth = 0;
  PathSegment* dev_path_end = dev_paths + pixelcount;
  int num_paths = dev_path_end - dev_paths;

  const int all_path_count = num_paths;

  // --- PathSegment Tracing Stage ---
  // Shoot ray into scene, bounce between objects, push shading chunks

  bool iterationComplete = false;
  while (!iterationComplete)
  {
    // clean shading chunks
    hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    // tracing
    dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
    computeIntersections <<<numblocksPathSegmentTracing, blockSize1d>>>(
      depth,
      num_paths,
      dev_paths,
      dev_geoms,
      int(hst_scene->geoms.size()),
      dev_intersections,
      dev_triangles,
      dev_kd_nodes,
      dev_kd_triangles
    );
    checkCUDAError("trace one bounce");

#ifdef SORT_INTERSECTIONS_BY_MATERIAL_ID
    fillMaterialArray<<<numblocksPathSegmentTracing, blockSize1d>>>(num_paths, dev_path_material_ids, dev_path_indices, dev_intersections);
    thrust::sort_by_key(dev_thrust_path_material_ids, dev_thrust_path_material_ids + num_paths, dev_thrust_path_indices);
    reshufflePathSegments<<<numblocksPathSegmentTracing, blockSize1d>>>(num_paths, dev_path_indices, dev_intersections, dev_paths, dev_intersections_b, dev_paths_b);

    PathSegment* tempA = dev_paths;
    ShadeableIntersection* tempB = dev_intersections;
    dev_paths = dev_paths_b;
    dev_intersections = dev_intersections_b;
    dev_paths_b = tempA;
    dev_intersections_b = tempB;
#endif

    dev_thrust_paths = thrust::device_ptr<PathSegment>(dev_paths);

#ifdef FULL_LIGHTING_INTEGRATOR
    shadeRays<<<numblocksPathSegmentTracing, blockSize1d>>>(
      iter,
      traceDepth,
      num_paths,
      hst_scene->m_numLights,
      int(hst_scene->geoms.size()),
      dev_intersections,
      dev_paths,
      dev_materials,
      dev_geom_lights,
      dev_geoms,
      dev_texels,
      dev_imageInfo,
      dev_triangles,
      dev_kd_nodes,
      dev_kd_triangles
    );
#elif defined(DIRECT_LIGHTING_INTEGRATOR)
    shadeDirectLighting<<<numblocksPathSegmentTracing, blockSize1d>>>(
      iter,
      traceDepth,
      num_paths,
      hst_scene->m_numLights,
      hst_scene->geoms.size(),
      dev_intersections,
      dev_paths,
      dev_materials,
      dev_geom_lights,
      dev_geoms,
      dev_texels,
      dev_imageInfo,
      dev_triangles,
      dev_kd_nodes,
      dev_kd_triangles
      );
#endif
    checkCUDAError("Shade Error");

    const auto middleItr = thrust::partition(dev_thrust_paths, dev_thrust_paths + num_paths, IsValidPath());
    iterationComplete = dev_paths == middleItr.get();
    num_paths = middleItr.get() - dev_paths;
  }

  hipDeviceSynchronize();

  // Assemble this iteration and apply it to the image
  dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
  finalGather<<<numBlocksPixels, blockSize1d>>>(all_path_count, dev_image, dev_paths);

  ///////////////////////////////////////////////////////////////////////////

  // Send results to OpenGL buffer for rendering
  sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);

  // Retrieve image from GPU
  hipMemcpy(hst_scene->state.image.data(), dev_image,
             pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

  checkCUDAError("pathtrace");
}
