#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <chrono>
#include <ctime>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"
#include "../stream_compaction/efficient.h"

#define ERRORCHECK 1
#define CACHE_FIRST_ITERATION 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char *msg, const char *file, int line) {
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err) {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file) {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
    getchar();
#  endif
    exit(EXIT_FAILURE);
#endif
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

__host__ __device__
void concentricSampleDisk(float* newX, float* newY, thrust::default_random_engine &rng)
{
  // get the sample
  thrust::uniform_real_distribution<float> u01(0.0f, 1.0f);
  float x = u01(rng);
  float y = u01(rng);

  // remap to -1 to 1
  float xOffset = 2.f * x - 1.f;
  float yOffset = 2.f * y - 1.f;

  if (xOffset == 0 && yOffset == 0)
  {
    *newX = xOffset;
    *newY = yOffset;
  }

  float theta, r;
  if (std::abs(xOffset) > std::abs(yOffset))
  {
    r = xOffset;
    theta = (PI / 4.f) * (yOffset / xOffset);
  }
  else 
  {
    r = yOffset;
    theta = (PI / 2.f) - ((PI / 4.f) * (xOffset / yOffset));
  }

  *newX = r * std::cos(theta);
  *newY = r * std::sin(theta);
}

__host__ __device__
void modifyRayForDepthofField(Ray* ray, float aperture, float focalDist, thrust::default_random_engine &rng)
{
  float lensX, lensY;

  concentricSampleDisk(&lensX, &lensY, rng);
  lensX *= aperture;
  lensY *= aperture;
  
  float ft = focalDist / fabs(ray->direction.z);
  glm::vec3 pFocus = getPointOnRay((*ray), ft);
  ray->origin += glm::vec3(lensX, lensY, 0.0f);
  ray->direction = glm::normalize(pFocus - ray->origin);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
        int iter, glm::vec3* image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int) (pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int) (pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int) (pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

static Scene * hst_scene = NULL;
static glm::vec3 * dev_image = NULL;
static Geom * dev_geoms = NULL;
static Material * dev_materials = NULL;
static PathSegment * dev_paths = NULL;
static PathSegment * dev_paths_first_iter_cache = NULL;
static ShadeableIntersection * dev_intersections = NULL;
static PathSegment ** dev_paths_ptrs = NULL;
static int * dev_material_ids = NULL;
// TODO: static variables for device memory, any extra info you need, etc
// ...

void pathtraceInit(Scene *scene) {
    hst_scene = scene;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

  	hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));
    hipMalloc(&dev_paths_ptrs, pixelcount * sizeof(PathSegment*));
    if (CACHE_FIRST_ITERATION) { hipMalloc(&dev_paths_first_iter_cache, pixelcount * sizeof(PathSegment)); }

/*    for (Geom g : scene->geoms)
    {
      if (g.numTriangles > 0)
      {
 //       hipMalloc(&g.dev_triangles, g.numTriangles * sizeof(Triangle));
 //       checkCUDAError("malloc triangles");
//
 //       hipMemcpy(g.dev_triangles, g.triangles, g.numTriangles * sizeof(Triangle), hipMemcpyHostToDevice);
 //       checkCUDAError("rip memcpy tris");

      }
    }
    */
    hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
    hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

  	hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
  	hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

  	hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
  	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

  	hipMalloc(&dev_material_ids, pixelcount * sizeof(int));

    // TODO: initialize any extra device memeory you need

    checkCUDAError("pathtraceInit");
}

void pathtraceFree() {
    hipFree(dev_image);  // no-op if dev_image is null
  	hipFree(dev_paths);
  	hipFree(dev_geoms);
  	hipFree(dev_materials);
  	hipFree(dev_intersections);
    hipFree(dev_paths_ptrs);
    if (CACHE_FIRST_ITERATION) { hipFree(dev_paths_first_iter_cache); }
    hipFree(dev_material_ids);

    // TODO: clean up any extra device memory you created

    checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < cam.resolution.x && y < cam.resolution.y) {
		int index = x + (y * cam.resolution.x);
		PathSegment & segment = pathSegments[index];

    segment.ray.origin = cam.position;
    segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

		segment.ray.direction = glm::normalize(cam.view
			- cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f)
			- cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f)
			);

    thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, traceDepth);

    modifyRayForDepthofField(&segment.ray, 0.5, 10, rng);
		segment.pixelIndex = index;
		segment.remainingBounces = traceDepth;
	}
}

// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
	int depth
	, int num_paths
	, PathSegment ** pathSegments
	, Geom * geoms
	, int geoms_size
	, ShadeableIntersection * intersections
	)
{
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (path_index < num_paths)
	{
		PathSegment* pathSegment_ptr = pathSegments[path_index];

		float t;
		glm::vec3 intersect_point;
		glm::vec3 normal;
		float t_min = FLT_MAX;
		int hit_geom_index = -1;
		bool outside = true;

		glm::vec3 tmp_intersect;
		glm::vec3 tmp_normal;

		// naive parse through global geoms

		for (int i = 0; i < geoms_size; i++)
		{
			Geom & geom = geoms[i];
			if (geom.type == CUBE)
			{
				t = boxIntersectionTest(geom, pathSegment_ptr->ray, tmp_intersect, tmp_normal, outside);
			}
			else if (geom.type == SPHERE)
			{
				t = sphereIntersectionTest(geom, pathSegment_ptr->ray, tmp_intersect, tmp_normal, outside);
			}
      else if (geom.type == SWORD)
      {
 //       float temp_t = boxIntersectionTest(geom, pathSegment_ptr->ray, tmp_intersect, tmp_normal, outside);
 //       if (temp_t > 0.0f && t_min > temp_t && outside)
//        {
          glm::vec3 baryPosition;
          for (int j = 0; j < geom.numTriangles; ++j)
          {
            if (glm::intersectRayTriangle(pathSegment_ptr->ray.origin,
              pathSegment_ptr->ray.direction,
              geom.dev_triangles[j].v1,
              geom.dev_triangles[j].v2,
              geom.dev_triangles[j].v3,
              baryPosition))
            {
              tmp_normal = geom.dev_triangles[j].n;
              tmp_intersect = baryPosition;
              t = baryPosition.z;
              break;
            }
          }
 //       }
      }
			// TODO: add more intersection tests here... triangle? metaball? CSG?

			// Compute the minimum t from the intersection tests to determine what
			// scene geometry object was hit first.
			if (t > 0.0f && t_min > t)
			{
        if (outside)
        {
          t_min = t;
          hit_geom_index = i;
          intersect_point = tmp_intersect;
          normal = tmp_normal;
        }
			}
		}

		if (hit_geom_index == -1)
		{
			intersections[pathSegment_ptr->pixelIndex].t = -1.0f;
		}
		else
		{
			//The ray hits something
			intersections[pathSegment_ptr->pixelIndex].t = t_min;
			intersections[pathSegment_ptr->pixelIndex].materialId = geoms[hit_geom_index].materialid;
			intersections[pathSegment_ptr->pixelIndex].surfaceNormal = normal;
		}
	}
}

// LOOK: "fake" shader demonstrating what you might do with the info in
// a ShadeableIntersection, as well as how to use thrust's random number
// generator. Observe that since the thrust random number generator basically
// adds "noise" to the iteration, the image should start off noisy and get
// cleaner as more iterations are computed.
//
// Note that this shader does NOT do a BSDF evaluation!
// Your shaders should handle that - this can allow techniques such as
// bump mapping.
__global__ void shadeFakeMaterial (
  int iter
  , int depth
  , int num_paths
	, ShadeableIntersection * shadeableIntersections
	, PathSegment ** pathSegments
	, Material * materials
	)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_paths)
  {
    ShadeableIntersection intersection = shadeableIntersections[pathSegments[idx]->pixelIndex];
    if (intersection.t > 0.0f && pathSegments[idx]->remainingBounces > 0) { // if the intersection exists...
      // Set up the RNG
      // LOOK: this is how you use thrust's RNG! Please look at
      // makeSeededRandomEngine as well.
//      long ms = std::chrono::system_clock::now().time_since_epoch().count;
      thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, depth);
      thrust::uniform_real_distribution<float> u01(0, 1);

      Material material = materials[intersection.materialId];
      glm::vec3 materialColor = material.color;

      // If the material indicates that the object was a light, "light" the ray
      if (material.emittance > 0.0f) {
        pathSegments[idx]->color *= (materialColor * material.emittance);
        pathSegments[idx]->remainingBounces = 0;
      }
      // Otherwise, do some pseudo-lighting computation. This is actually more
      // like what you would expect from shading in a rasterizer like OpenGL.
      // TODO: replace this! you should be able to start with basically a one-liner
      else {
        float lightTerm = glm::dot(intersection.surfaceNormal, glm::vec3(0.0f, 1.0f, 0.0f));
        pathSegments[idx]->color *= (materialColor * lightTerm) * 0.3f + ((1.0f - intersection.t * 0.02f) * materialColor) * 0.7f;
        pathSegments[idx]->color *= u01(rng); // apply some noise because why not
        pathSegments[idx]->remainingBounces--;
      }
    // If there was no intersection, color the ray black.
    // Lots of renderers use 4 channel color, RGBA, where A = alpha, often
    // used for opacity, in which case they can indicate "no opacity".
    // This can be useful for post-processing and image compositing.
    } else {
      pathSegments[idx]->color = glm::vec3(0.0f);
      pathSegments[idx]->remainingBounces = 0;
    }
  }
}


__global__ void shadeRealMaterial (
  int iter
  , int depth
  , int num_paths
	, ShadeableIntersection * shadeableIntersections
	, PathSegment ** pathSegments
	, Material * materials
	)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_paths)
  {
    ShadeableIntersection intersection = shadeableIntersections[pathSegments[idx]->pixelIndex];
    
    if (intersection.t > 0.0f && pathSegments[idx]->remainingBounces > 0) { // if the intersection exists...
      // Set up the RNG
      // LOOK: this is how you use thrust's RNG! Please look at
      // makeSeededRandomEngine as well.
      thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, depth);
      thrust::uniform_real_distribution<float> u01(0.0f, 1.0f);

      Material material = materials[intersection.materialId];
      glm::vec3 materialColor = material.color;

      // If the material indicates that the object was a light, "light" the ray
      if (material.emittance > 0.0f) {
        pathSegments[idx]->color *= (materialColor * material.emittance);
        pathSegments[idx]->remainingBounces = 0;
      }
      // Otherwise, do some pseudo-lighting computation. This is actually more
      // like what you would expect from shading in a rasterizer like OpenGL.
      // TODO: replace this! you should be able to start with basically a one-liner
      else {
        glm::vec3 intersectionPoint = getPointOnRay(pathSegments[idx]->ray, intersection.t);
        scatterRay(pathSegments[idx], intersection.t, intersectionPoint, intersection.surfaceNormal, material, rng);
        pathSegments[idx]->remainingBounces--;
      }
    // If there was no intersection, color the ray black.
    // Lots of renderers use 4 channel color, RGBA, where A = alpha, often
    // used for opacity, in which case they can indicate "no opacity".
    // This can be useful for post-processing and image compositing.
    } else {
      pathSegments[idx]->color = glm::vec3(0.0f);
      pathSegments[idx]->remainingBounces = 0;
    }
  }
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3 * image, PathSegment * iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < nPaths)
	{
		PathSegment iterationPath = iterationPaths[index];
		image[iterationPath.pixelIndex] += iterationPath.color;
	}
}

// very simple function to get pointers to all the paths to prevent having to copy so much during stream compaction
__global__ void getPointersToPaths(int nPaths, PathSegment** dev_paths_ptrs, PathSegment* dev_paths)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < nPaths)
  {
    dev_paths_ptrs[idx] = &(dev_paths[idx]);
  }
}


// very simple kernel to set up our thrust sort
__global__ void getMaterialIDArray(int nPaths, int* dev_materialIDs, 
  ShadeableIntersection* dev_intersections, PathSegment** dev_paths_ptrs)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < nPaths)
  {
    dev_materialIDs[idx] = dev_intersections[dev_paths_ptrs[idx]->pixelIndex].materialId;
  }
}

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4 *pbo, int frame, int iter) {
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

	// 2D block for generating ray from camera
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
            (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
            (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

	// 1D block for path tracing
	const int blockSize1d = 128;

    ///////////////////////////////////////////////////////////////////////////

    // Recap:
    // * Initialize array of path rays (using rays that come out of the camera)
    //   * You can pass the Camera object to that kernel.
    //   * Each path ray must carry at minimum a (ray, color) pair,
    //   * where color starts as the multiplicative identity, white = (1, 1, 1).
    //   * This has already been done for you.
    // * For each depth:
    //   * Compute an intersection in the scene for each path ray.
    //     A very naive version of this has been implemented for you, but feel
    //     free to add more primitives and/or a better algorithm.
    //     Currently, intersection distance is recorded as a parametric distance,
    //     t, or a "distance along the ray." t = -1.0 indicates no intersection.
    //     * Color is attenuated (multiplied) by reflections off of any object
    //   * TODO: Stream compact away all of the terminated paths.
    //     You may use either your implementation or `thrust::remove_if` or its
    //     cousins.
    //     * Note that you can't really use a 2D kernel launch any more - switch
    //       to 1D.
    //   * TODO: Shade the rays that intersected something or didn't bottom out.
    //     That is, color the ray by performing a color computation according
    //     to the shader, then generate a new ray to continue the ray path.
    //     We recommend just updating the ray's PathSegment in place.
    //     Note that this step may come before or after stream compaction,
    //     since some shaders you write may also cause a path to terminate.
    // * Finally, add this iteration's results to the image. This has been done
    //   for you.

  if (CACHE_FIRST_ITERATION)
  {
    // save the very first iteration into the other buffer
    if (iter == 1)
    {
      generateRayFromCamera << <blocksPerGrid2d, blockSize2d >> > (cam, iter, traceDepth, dev_paths_first_iter_cache);
      checkCUDAError("generate camera ray");
    }

    // memcpy the cache buffer into the dev_paths buffer
    hipMemcpy(dev_paths, dev_paths_first_iter_cache, pixelcount * sizeof(PathSegment), hipMemcpyDeviceToDevice);
  }
  else
  {
    // if we aren't caching then just generate rays into dev_paths always
    generateRayFromCamera << <blocksPerGrid2d, blockSize2d >> > (cam, iter, traceDepth, dev_paths);
    checkCUDAError("generate camera ray");
  }

	int depth = 0;
	PathSegment* dev_path_end = dev_paths + pixelcount;
	int num_paths = dev_path_end - dev_paths;

	// --- PathSegment Tracing Stage ---
	// Shoot ray into scene, bounce between objects, push shading chunks
  dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
  getPointersToPaths << <numblocksPathSegmentTracing, blockSize1d >> > (num_paths, dev_paths_ptrs, dev_paths);

  bool iterationComplete = false;
	while (!iterationComplete) {

    // clean shading chunks
    hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    // tracing
    numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
    computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (depth, num_paths, dev_paths_ptrs, dev_geoms, hst_scene->geoms.size(), dev_intersections);
    checkCUDAError("trace one bounce");
    hipDeviceSynchronize();
    depth++;

    /* 
    --- Shading Stage ---
    Shade path segments based on intersections and generate new rays by evaluating the BSDF.
    Start off with just a big kernel that handles all the different materials you have in the scenefile. 
    */

    thrust::device_ptr<int> dev_materialIDs_thrust(dev_material_ids);
    thrust::device_ptr<PathSegment*> dev_paths_thrust(dev_paths_ptrs);
    getMaterialIDArray << <numblocksPathSegmentTracing, blockSize1d >> > (num_paths, dev_material_ids, dev_intersections, dev_paths_ptrs);
    thrust::sort_by_key(dev_materialIDs_thrust, dev_materialIDs_thrust + num_paths, dev_paths_thrust);

    shadeRealMaterial << <numblocksPathSegmentTracing, blockSize1d >> > (iter, depth, num_paths, dev_intersections, dev_paths_ptrs, dev_materials);
    checkCUDAError("shade real material");

    // now we call the stream compaction
    num_paths = StreamCompaction::Efficient::compact(num_paths, dev_paths_ptrs, dev_paths_ptrs);
    checkCUDAError("Stream Compaction");

    if (num_paths <= 0)
    {
      iterationComplete = true;
    }
 	}

  // Assemble this iteration and apply it to the image
  dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
	finalGather<<<numBlocksPixels, blockSize1d>>>(pixelcount, dev_image, dev_paths);

  ///////////////////////////////////////////////////////////////////////////

  // Send results to OpenGL buffer for rendering
  sendImageToPBO << <blocksPerGrid2d, blockSize2d >> > (pbo, cam.resolution, iter, dev_image);

  // Retrieve image from GPU
  hipMemcpy(hst_scene->state.image.data(), dev_image, pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);
  checkCUDAError("pathtrace");
}
