#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/partition.h>
#include <thrust/device_ptr.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"
#include "warpfunctions.h"
#include "materials.h"
#include "lights.h"

// #define SORT_INTERSECTIONS_BY_MATERIAL_ID



#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)

void checkCUDAErrorFn(const char* msg, const char* file, int line)
{
#if ERRORCHECK
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (hipSuccess == err)
  {
    return;
  }

  fprintf(stderr, "CUDA error");
  if (file)
  {
    fprintf(stderr, " (%s:%d)", file, line);
  }
  fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
  getchar();
#  endif
  exit(EXIT_FAILURE);
#endif
}

__host__ __device__

thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth)
{
  int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
  return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
                               int iter, glm::vec3* image)
{
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;

  if (x < resolution.x && y < resolution.y)
  {
    int index = x + (y * resolution.x);
    glm::vec3 pix = image[index];

    glm::ivec3 color;
    color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
    color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
    color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);

    // Each thread writes one pixel location in the texture (textel)
    pbo[index].w = 0;
    pbo[index].x = color.x;
    pbo[index].y = color.y;
    pbo[index].z = color.z;
  }
}

static Scene* hst_scene = NULL;
static glm::vec3* dev_image = NULL;
static Geom* dev_geoms = NULL;
static Geom* dev_geom_lights = NULL;
static Material* dev_materials = NULL;
static int* dev_path_material_ids = NULL;
static int* dev_path_indices = NULL;
static PathSegment* dev_paths = NULL;
static PathSegment* dev_paths_b = NULL;
static ImageInfo* dev_imageInfo = NULL;
static glm::vec3* dev_texels = NULL;
thrust::device_ptr<PathSegment> dev_thrust_paths;
static ShadeableIntersection* dev_intersections = NULL;
static ShadeableIntersection* dev_intersections_b = NULL;
static thrust::device_ptr<int> dev_thrust_path_indices;
static thrust::device_ptr<int> dev_thrust_path_material_ids;
// TODO: static variables for device memory, any extra info you need, etc
// ...

void pathtraceInit(Scene* scene)
{
  hst_scene = scene;
  const Camera& cam = hst_scene->state.camera;
  const int pixelcount = cam.resolution.x * cam.resolution.y;

  hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
  hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

  hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));
  hipMalloc(&dev_paths_b, pixelcount * sizeof(PathSegment));
  hipMalloc(&dev_path_material_ids, pixelcount * sizeof(int));
  hipMalloc(&dev_path_indices, pixelcount * sizeof(int));

  dev_thrust_path_material_ids = thrust::device_ptr<int>(dev_path_material_ids);
  dev_thrust_path_indices = thrust::device_ptr<int>(dev_path_indices);

  hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
  hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);
  
  hipMalloc(&dev_imageInfo, scene->imageInfo.size() * sizeof(ImageInfo));
  hipMemcpy(dev_imageInfo, scene->imageInfo.data(), scene->imageInfo.size() * sizeof(ImageInfo), hipMemcpyHostToDevice);

  hipMalloc(&dev_texels, scene->allTexels.size() * sizeof(glm::vec3));
  hipMemcpy(dev_texels, scene->allTexels.data(), scene->allTexels.size() * sizeof(glm::vec3), hipMemcpyHostToDevice);
  
  std::vector<Geom> lights;
  for(const auto& geom : scene->geoms)
  {
    if (scene->materials[geom.materialid].emittance <= 0)
    {
      continue;
    }

    lights.push_back(geom);
  }

  scene->m_numLights = lights.size();

  hipMalloc(&dev_geom_lights, lights.size() * sizeof(Geom));
  hipMemcpy(dev_geom_lights, lights.data(), lights.size() * sizeof(Geom), hipMemcpyHostToDevice);

  hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
  hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material),
             hipMemcpyHostToDevice);

  hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
  hipMalloc(&dev_intersections_b, pixelcount * sizeof(ShadeableIntersection));
  hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));
  hipMemset(dev_intersections_b, 0, pixelcount * sizeof(ShadeableIntersection));

  checkCUDAError("pathtraceInit");
}

void pathtraceFree()
{
  hipFree(dev_image); // no-op if dev_image is null
  hipFree(dev_paths);
  hipFree(dev_geoms);
  hipFree(dev_materials);
  hipFree(dev_intersections);
  hipFree(dev_geom_lights);
  hipFree(dev_path_material_ids);
  hipFree(dev_path_indices);
  hipFree(dev_paths_b);
  hipFree(dev_imageInfo);
  hipFree(dev_texels);
  hipFree(dev_intersections_b);

  checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;

  if (x < cam.resolution.x && y < cam.resolution.y)
  {
    int index = x + (y * cam.resolution.x);
    PathSegment& segment = pathSegments[index];

    segment.ray.origin = cam.position;
    segment.color = glm::vec3(0.0f, 0.0f, 0.0f);
    segment.throughput = glm::vec3(1.0f, 1.0f, 1.0f);

    // TODO: implement antialiasing by jittering the ray
    segment.ray.direction = glm::normalize(cam.view
      - cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f)
      - cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f)
    );

    segment.pixelIndex = index;
    segment.remainingBounces = traceDepth;
    segment.rayFromSpecular = false;
  }
}

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
  int depth
  , int num_paths
  , PathSegment* pathSegments
  , Geom* geoms
  , int geoms_size
  , ShadeableIntersection* intersections
)
{
  int path_index = blockIdx.x * blockDim.x + threadIdx.x;

  if (path_index < num_paths)
  {
    const PathSegment& pathSegment = pathSegments[path_index];
    int pixelIndex = pathSegment.pixelIndex;
    intersections[path_index] = Intersections::Do(pathSegment.ray, geoms, geoms_size);
  }
}

__device__ float PowerHeuristic(int nf, Float fPdf, int ng, Float gPdf)
{
  Float f = nf * fPdf;
  Float g = ng * gPdf;
  return (f * f) / (f * f + g * g);
}


__device__ Color3f GetTextureData(const ImageInfo& info, const glm::vec2& uv, glm::vec3* texels)
{
  int startTexelIdx = info.startIdx;

  int X = glm::min(info.width * uv.x, info.width - 1.0f);
  int Y = glm::min(info.height * uv.y, info.height - 1.0f); // flipped from stb image
  const int coord = startTexelIdx + X + info.height * Y;
  return texels[coord];
}

// LOOK: "fake" shader demonstrating what you might do with the info in
// a ShadeableIntersection, as well as how to use thrust's random number
// generator. Observe that since the thrust random number generator basically
// adds "noise" to the iteration, the image should start off noisy and get
// cleaner as more iterations are computed.
//
// Note that this shader does NOT do a BSDF evaluation!
// Your shaders should handle that - this can allow techniques such as
// bump mapping.
__global__ void shadeRays(
  int iter,
  int maxDepth,
  int num_paths,
  int num_lights,
  int geoms_size,
  ShadeableIntersection* shadeableIntersections,
  PathSegment* pathSegments,
  Material* materials,
  Geom* lights,
  Geom* geoms,
  glm::vec3* texels,
  ImageInfo* imageInfos
)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= num_paths)
  {
    return;
  }

  PathSegment& targetSegment = pathSegments[idx];
  const int pixelIndex = targetSegment.pixelIndex;

  // Didn't hit anything or hit something behind
  const ShadeableIntersection intersection = shadeableIntersections[idx];
  if (intersection.t <= 0.0f)
  {
    targetSegment.remainingBounces = 0;
    return;
  }

  // if the intersection exists...
  // Set up the RNG
  // LOOK: this is how you use thrust's RNG! Please look at
  // makeSeededRandomEngine as well.
  thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);
  thrust::uniform_real_distribution<float> u01(0, 1);

  const Material material = materials[intersection.materialId];
  const bool sampledSpecular = material.type == SPECULAR || material.type == ROUGH_SPECULAR;

  // If the material indicates that the object was a light, "light" the ray
  if (material.emittance > 0.0f)
  {
    // Max Depth - Hit Light Directly
    if (maxDepth == pathSegments[idx].remainingBounces || targetSegment.rayFromSpecular)
    {
      targetSegment.color += (targetSegment.throughput * material.color * material.emittance);
    }

    // Terminate Ray
    targetSegment.remainingBounces = 0;
    return;
  }

  const glm::vec3 woW = -targetSegment.ray.direction;
  const glm::vec3 wo = intersection.worldToTangent * woW;
  glm::vec3 directWiW;
  float directPdf = 0.0f;

  glm::vec3 indirectWi;
  glm::vec3 indirectWiW = glm::vec3(pixelIndex);
  float indirectPdf = 0.0f;
  Color3f indirectFrTerm;

  Color3f finalColor = Color3f(0.0f);

  const int randomIdx = (int)(u01(rng) * num_lights);
  Geom* activeLight = &lights[randomIdx];
  const Material lightMaterial = materials[activeLight->materialid];

  Color3f diffuseMaterialColor = material.color;
  if (material.diffuseMapId >= 0)
  {
    diffuseMaterialColor = GetTextureData(imageInfos[material.diffuseMapId], intersection.uv, texels);
  }

  Intersection lightIntr;

  float directFactor = 0.0f;
  float indirectFactor = 0.0f;

  if (!sampledSpecular) {

    if (material.type == DIFFUSE)
    {
      indirectFrTerm = BRDF::Lambert::Sample_f(diffuseMaterialColor, wo, &indirectWi, &indirectPdf, u01(rng), u01(rng));
    }

    indirectWiW = intersection.tangentToWorld * indirectWi;

    const Color3f directLi = Lights::Arealight::Sample_Li(lightMaterial.color * lightMaterial.emittance, intersection.intersectPoint, u01(rng), u01(rng), activeLight, &directWiW, &directPdf, &lightIntr);
    directPdf = directPdf / static_cast<float>(num_lights);

    if (directPdf > EPSILON)
    {
      const Ray shadowRay = Intersections::SpawnRay(intersection.intersectPoint, intersection.surfaceNormal, directWiW);
      const auto shadowIntr = Intersections::Do(shadowRay, geoms, geoms_size);

      if (shadowIntr.geom != nullptr)
      {
        // ID compare
        if (shadowIntr.geom->id == activeLight->id)
        {
          const float directCosTerm = std::abs(glm::dot(intersection.surfaceNormal, directWiW));
          const glm::vec3 directWi = intersection.worldToTangent * directWiW;

          if (material.type == DIFFUSE)
          {
            const Color3f directFrTerm = BRDF::Lambert::f(diffuseMaterialColor, wo, directWi);
            directFactor = PowerHeuristic(1, directPdf, 1, BRDF::Lambert::Pdf(wo, directWi));
            finalColor += ((directFrTerm * directLi * directCosTerm * directFactor) / directPdf);
          }
        }
      }
    }

    if (indirectPdf > EPSILON)
    {
      float lightPdf = Lights::Arealight::Pdf_Li(intersection.intersectPoint, intersection.surfaceNormal, indirectWiW, activeLight);
      if (lightPdf > EPSILON) {
        lightPdf = lightPdf / num_lights;
        indirectFactor = PowerHeuristic(1, indirectPdf, 1, lightPdf);
      }

      Ray indirectRay = Intersections::SpawnRay(intersection.intersectPoint, intersection.surfaceNormal, indirectWiW);
      Intersection indirectIsect;

      const float indirectCosTerm = std::abs(glm::dot(intersection.surfaceNormal, indirectWiW));

      const auto indirectIntr = Intersections::Do(indirectRay, geoms, geoms_size);

      Color3f indirectLiTerm = Color3f(0.0f);

      if (indirectIntr.geom != nullptr)
      {
        if (indirectIntr.geom->id == activeLight->id) {
          indirectLiTerm = Lights::Arealight::L(lightMaterial.color * lightMaterial.emittance, indirectIntr.surfaceNormal, -indirectWiW);
        }

        finalColor += ((indirectFrTerm * indirectLiTerm * indirectCosTerm * indirectFactor) / indirectPdf);
      }
    }
  }

  // Add MIS Color
  targetSegment.color += (finalColor * targetSegment.throughput);

  targetSegment.remainingBounces--;

  if (targetSegment.remainingBounces <= 0)
  {
    // No Need to compute next ray
    return;
  }

  Vector3f bounceWi;
  Vector3f bounceWiW;
  float bouncePdf;
  Color3f bounceFrTerm;

  targetSegment.rayFromSpecular = false;

  if (material.type == DIFFUSE)
  {
    bounceFrTerm = BRDF::Lambert::Sample_f(diffuseMaterialColor, wo, &bounceWi, &bouncePdf, u01(rng), u01(rng));
  }
  else if (material.type == SPECULAR)
  {
    bounceFrTerm = BRDF::Specular::Sample_f(diffuseMaterialColor, wo, &bounceWi, &bouncePdf, FRESNEL_NOOP);
    targetSegment.rayFromSpecular = true;
  }
  else if (material.type == ROUGH_SPECULAR)
  {
    bounceFrTerm = BRDF::Microfacet::Sample_f(material.color, wo, &bounceWi, &bouncePdf, FRESNEL_NOOP, u01(rng), u01(rng), material.roughness, material.roughness);
    targetSegment.rayFromSpecular = true;
  }

  bounceWiW = intersection.tangentToWorld * bounceWi;

  const float bounceCosTerm = std::abs(glm::dot(intersection.surfaceNormal, bounceWiW));

  if (bouncePdf < EPSILON) {
    // Terminate Ray
    targetSegment.remainingBounces = 0;
    return;
  }

  targetSegment.throughput = (targetSegment.throughput * bounceFrTerm * bounceCosTerm) / bouncePdf;
  targetSegment.ray = Intersections::SpawnRay(intersection.intersectPoint, intersection.surfaceNormal, bounceWiW);

  // Russian Roulette
  const float maxVal = glm::max(glm::max(static_cast<float>(targetSegment.throughput[0]), targetSegment.throughput[1]), targetSegment.throughput[2]);

  if (u01(rng) < (1.0f - maxVal)) {
    targetSegment.remainingBounces = 0;
    return;
  }
  
  targetSegment.throughput /= maxVal;
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3* image, PathSegment* iterationPaths)
{
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (index < nPaths)
  {
    PathSegment iterationPath = iterationPaths[index];
    image[iterationPath.pixelIndex] += iterationPath.color;
  }
}

__global__ void fillMaterialArray(int nPaths, int* materialIds, int* pathIndices, ShadeableIntersection* shadeableIntersections)
{
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (index < nPaths)
  {
    const ShadeableIntersection& shadeIntr = shadeableIntersections[index];
    materialIds[index] = shadeIntr.materialId;
    pathIndices[index] = index;
  }
}

__global__ void reshufflePathSegments(int nPaths, int* pathIndices, ShadeableIntersection* shadeableIntersections, PathSegment* segments,  ShadeableIntersection* oIntrs, PathSegment* oSegments)
{
  const int index = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (index < nPaths)
  {
    const int idx = pathIndices[index];
    oIntrs[index] = shadeableIntersections[idx];
    oSegments[index] = segments[idx];
  }
}


struct IsValidPath
{
  __host__ __device__ bool operator() (const PathSegment& segment)
  {
    return segment.remainingBounces > 0;
  }
};

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4* pbo, int frame, int iter)
{
  const int traceDepth = hst_scene->state.traceDepth;
  const Camera& cam = hst_scene->state.camera;
  const int pixelcount = cam.resolution.x * cam.resolution.y;

  // 2D block for generating ray from camera
  const dim3 blockSize2d(8, 8);
  const dim3 blocksPerGrid2d(
    (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
    (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

  // 1D block for path tracing
  const int blockSize1d = 256;

  ///////////////////////////////////////////////////////////////////////////

  // Recap:
  // * Initialize array of path rays (using rays that come out of the camera)
  //   * You can pass the Camera object to that kernel.
  //   * Each path ray must carry at minimum a (ray, color) pair,
  //   * where color starts as the multiplicative identity, white = (1, 1, 1).
  //   * This has already been done for you.
  // * For each depth:
  //   * Compute an intersection in the scene for each path ray.
  //     A very naive version of this has been implemented for you, but feel
  //     free to add more primitives and/or a better algorithm.
  //     Currently, intersection distance is recorded as a parametric distance,
  //     t, or a "distance along the ray." t = -1.0 indicates no intersection.
  //     * Color is attenuated (multiplied) by reflections off of any object
  //   * TODO: Stream compact away all of the terminated paths.
  //     You may use either your implementation or `thrust::remove_if` or its
  //     cousins.
  //     * Note that you can't really use a 2D kernel launch any more - switch
  //       to 1D.
  //   * TODO: Shade the rays that intersected something or didn't bottom out.
  //     That is, color the ray by performing a color computation according
  //     to the shader, then generate a new ray to continue the ray path.
  //     We recommend just updating the ray's PathSegment in place.
  //     Note that this step may come before or after stream compaction,
  //     since some shaders you write may also cause a path to terminate.
  // * Finally, add this iteration's results to the image. This has been done
  //   for you.

  // TODO: perform one iteration of path tracing

  generateRayFromCamera <<<blocksPerGrid2d, blockSize2d >>>(cam, iter, traceDepth, dev_paths);
  checkCUDAError("generate camera ray");

  int depth = 0;
  PathSegment* dev_path_end = dev_paths + pixelcount;
  int num_paths = dev_path_end - dev_paths;

  const int all_path_count = num_paths;

  // --- PathSegment Tracing Stage ---
  // Shoot ray into scene, bounce between objects, push shading chunks

  bool iterationComplete = false;
  while (!iterationComplete)
  {
    // clean shading chunks
    hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    // tracing
    dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
    computeIntersections <<<numblocksPathSegmentTracing, blockSize1d>>>(
      depth,
      num_paths,
      dev_paths,
      dev_geoms,
      hst_scene->geoms.size(),
      dev_intersections
    );
    checkCUDAError("trace one bounce");

#ifdef SORT_INTERSECTIONS_BY_MATERIAL_ID
    fillMaterialArray<<<numblocksPathSegmentTracing, blockSize1d>>>(num_paths, dev_path_material_ids, dev_path_indices, dev_intersections);
    thrust::sort_by_key(dev_thrust_path_material_ids, dev_thrust_path_material_ids + num_paths, dev_thrust_path_indices);
    reshufflePathSegments<<<numblocksPathSegmentTracing, blockSize1d>>>(num_paths, dev_path_indices, dev_intersections, dev_paths, dev_intersections_b, dev_paths_b);

    PathSegment* tempA = dev_paths;
    ShadeableIntersection* tempB = dev_intersections;
    dev_paths = dev_paths_b;
    dev_intersections = dev_intersections_b;
    dev_paths_b = tempA;
    dev_intersections_b = tempB;
#endif

    dev_thrust_paths = thrust::device_ptr<PathSegment>(dev_paths);

    shadeRays<<<numblocksPathSegmentTracing, blockSize1d>>>(
      iter,
      traceDepth,
      num_paths,
      hst_scene->m_numLights,
      hst_scene->geoms.size(),
      dev_intersections,
      dev_paths,
      dev_materials,
      dev_geom_lights,
      dev_geoms,
      dev_texels,
      dev_imageInfo
    );

    const auto middleItr = thrust::partition(dev_thrust_paths, dev_thrust_paths + num_paths, IsValidPath());
    iterationComplete = dev_paths == middleItr.get();
    num_paths = middleItr.get() - dev_paths;
  }

  hipDeviceSynchronize();

  // Assemble this iteration and apply it to the image
  dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
  finalGather<<<numBlocksPixels, blockSize1d>>>(all_path_count, dev_image, dev_paths);

  ///////////////////////////////////////////////////////////////////////////

  // Send results to OpenGL buffer for rendering
  sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);

  // Retrieve image from GPU
  hipMemcpy(hst_scene->state.image.data(), dev_image,
             pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

  checkCUDAError("pathtrace");
}
